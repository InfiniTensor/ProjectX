﻿#include "functions.cuh"

namespace refactor::hardware {

    void setDevice(int device) {
        CUDA_ASSERT(hipSetDevice(device));
    }
    int getDeviceCount() {
        int deviceCount;
        CUDA_ASSERT(hipGetDeviceCount(&deviceCount));
        return deviceCount;
    }

}// namespace refactor::hardware
