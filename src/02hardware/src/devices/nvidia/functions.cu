﻿#include "functions.cuh"

namespace refactor::hardware {

    void setDevice(int device) {
        CUDA_ASSERT(hipSetDevice(device));
    }

}// namespace refactor::hardware
