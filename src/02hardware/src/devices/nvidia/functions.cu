﻿#include "functions.cuh"

namespace refactor::hardware {

    int getDeviceCount() {
        int deviceCount;
        CUDA_ASSERT(hipGetDeviceCount(&deviceCount));
        return deviceCount;
    }
    void setDevice(int device) {
        CUDA_ASSERT(hipSetDevice(device));
    }
    MemInfo getMemInfo() {
        MemInfo memInfo;
        CUDA_ASSERT(hipMemGetInfo(&memInfo.free, &memInfo.total));
        return memInfo;
    }

}// namespace refactor::hardware
