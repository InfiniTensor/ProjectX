﻿#include "functions.cuh"
#include "memory.cuh"

namespace refactor::hardware {
    using M = NvidiaMemory;

    void *M::malloc(size_t size) noexcept {
        void *ptr;
        CUDA_ASSERT(hipMalloc(&ptr, size));
        return ptr;
    }
    void M::free(void *ptr) noexcept {
        if (auto status = hipFree(ptr); status != hipSuccess && status != hipErrorDeinitialized) {
            RUNTIME_ERROR(fmt::format("hipFree failed with \"{}\" ({})",
                                      hipGetErrorString(status), (int) status));
        }
    }
    void *M::copyHD(void *dst, void const *src, size_t bytes) const noexcept {
        CUDA_ASSERT(hipMemcpy(dst, src, bytes, hipMemcpyHostToDevice));
    }
    void *M::copyDH(void *dst, void const *src, size_t bytes) const noexcept {
        CUDA_ASSERT(hipMemcpy(dst, src, bytes, hipMemcpyDeviceToHost));
    }
    void *M::copyDD(void *dst, void const *src, size_t bytes) const noexcept {
        CUDA_ASSERT(hipMemcpy(dst, src, bytes, hipMemcpyDeviceToDevice));
    }

}// namespace refactor::hardware
