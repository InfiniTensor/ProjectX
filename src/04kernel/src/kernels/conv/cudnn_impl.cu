﻿#include "../../utilities/cuda/cudnn_context.hh"
#include "../../utilities/cuda/cudnn_functions.h"
#include "cudnn_kernel.hh"
#include "runtime/mem_manager.hh"

namespace refactor::kernel {
    using namespace cudnn;
    using namespace runtime;

    auto ConvCudnn::lower() const noexcept -> Routine {
        // RAII for closure
        struct Descriptors {
            hipdnnTensorDescriptor_t x, y;
            hipdnnFilterDescriptor_t w;
            hipdnnConvolutionDescriptor_t conv;
            hipdnnConvolutionFwdAlgo_t algo;

            Descriptors() {
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&x));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&y));
                CUDNN_ASSERT(hipdnnCreateFilterDescriptor(&w));
                CUDNN_ASSERT(hipdnnCreateConvolutionDescriptor(&conv));
            }
            ~Descriptors() {
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(x));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(y));
                CUDNN_ASSERT(hipdnnDestroyFilterDescriptor(w));
                CUDNN_ASSERT(hipdnnDestroyConvolutionDescriptor(conv));
            }

            Descriptors(const Descriptors &) = delete;
            Descriptors(Descriptors &&) = delete;
        };
        auto d = std::make_shared<Descriptors>();

        d->algo = static_cast<hipdnnConvolutionFwdAlgo_t>(info.algo);
        auto cudnnDataType = cudnnDataTypeConvert(info.dt);
        auto xs = info.xShape, ys = info.yShape, ws = info.wShape;
        CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(d->x, HIPDNN_TENSOR_NCHW, cudnnDataType, xs[0], xs[1], xs[2], xs[3]));
        CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(d->y, HIPDNN_TENSOR_NCHW, cudnnDataType, ys[0], ys[1], ys[2], ys[3]));
        CUDNN_ASSERT(hipdnnSetFilter4dDescriptor(d->w, cudnnDataType, HIPDNN_TENSOR_NCHW, ws[0], ws[1], ws[2], ws[3]));
        auto pp = info.pad;
        auto ss = info.stride;
        auto dd = info.dilation;
        CUDNN_ASSERT(hipdnnSetConvolution2dDescriptor(d->conv, pp[0], pp[1], ss[0], ss[1], dd[0], dd[1], HIPDNN_CROSS_CORRELATION, cudnnDataType));

        // nvcc at c++11 doesn't support real move capture
        return [d_ = std::move(d)](Resources &res, void const **inputs, void **outputs) {
            // fetch cudnn handle from resources
            auto handle = res.fetchOrStore<CudnnContext>()->handle;
            constexpr static auto workspaceSize = 1ul << 30;
            auto workspace = mem_manager::ForeignBlob::share(res.fetch<MemManager>()->manager, workspaceSize);
            auto const &d = *d_;
            // name inputs and outputs
            auto x = inputs[0],
                 w = inputs[1];
            auto y = outputs[0];
            // TODO? build alpha/beta for double
            float alpha = 1, beta = 0;
            CUDNN_ASSERT(hipdnnConvolutionForward(
                handle,
                &alpha,
                d.x, x,
                d.w, w,
                d.conv, d.algo,
                *workspace, workspaceSize,
                &beta,
                d.y, y));
        };
    }

}// namespace refactor::kernel
