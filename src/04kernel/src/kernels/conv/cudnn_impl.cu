﻿#include "../../utilities/cuda/cudnn_context.hh"
#include "../../utilities/cuda/cudnn_functions.h"
#include "../../utilities/cuda/cudnn_workspace.hh"
#include "cudnn_impl.h"

namespace refactor::kernel::cudnn {
    using namespace runtime;

    Routine ConvInfo::lower() const {
        // RAII for closure
        struct Descriptors {
            hipdnnTensorDescriptor_t x, y;
            hipdnnFilterDescriptor_t w;
            hipdnnConvolutionDescriptor_t conv;
            hipdnnConvolutionFwdAlgo_t algo;

            Descriptors() {
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&x));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&y));
                CUDNN_ASSERT(hipdnnCreateFilterDescriptor(&w));
                CUDNN_ASSERT(hipdnnCreateConvolutionDescriptor(&conv));
            }
            ~Descriptors() {
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(x));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(y));
                CUDNN_ASSERT(hipdnnDestroyFilterDescriptor(w));
                CUDNN_ASSERT(hipdnnDestroyConvolutionDescriptor(conv));
            }

            Descriptors(const Descriptors &) = delete;
            Descriptors(Descriptors &&) = delete;
        };
        auto d = std::make_shared<Descriptors>();

        d->algo = static_cast<hipdnnConvolutionFwdAlgo_t>(algo);
        auto cudnnDataType = cudnnDataTypeConvert(dt);
        CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(d->x, HIPDNN_TENSOR_NCHW, cudnnDataType, xShape[0], xShape[1], xShape[2], xShape[3]));
        CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(d->y, HIPDNN_TENSOR_NCHW, cudnnDataType, yShape[0], yShape[1], yShape[2], yShape[3]));
        CUDNN_ASSERT(hipdnnSetFilter4dDescriptor(d->w, cudnnDataType, HIPDNN_TENSOR_NCHW, wShape[0], wShape[1], wShape[2], wShape[3]));
        CUDNN_ASSERT(hipdnnSetConvolution2dDescriptor(d->conv, pad[0], pad[1], stride[0], stride[1], dilation[0], dilation[1], HIPDNN_CROSS_CORRELATION, cudnnDataType));

        // nvcc at c++11 doesn't support real move capture
        return [d_ = std::move(d)](Resources &res, void const **inputs, void **outputs) {
            // fetch cudnn handle from resources
            auto handle = res.fetchOrStore<CudnnContext>()->handle;
            auto const &workspace = *res.fetchOrStore<CudnnWorkspace>();
            auto const &d = *d_;
            // name inputs and outputs
            auto x = inputs[0],
                 w = inputs[1];
            auto y = outputs[0];
            // TODO? build alpha/beta for double
            float alpha = 1, beta = 0;
            CUDNN_ASSERT(hipdnnConvolutionForward(
                handle,
                &alpha,
                d.x, x,
                d.w, w,
                d.conv, d.algo,
                workspace.ptr, workspace.size,
                &beta,
                d.y, y));
        };
    }

}// namespace refactor::kernel::cudnn
