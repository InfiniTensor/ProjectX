﻿#include "../../utilities/cuda/cudnn_context.hh"
#include "../../utilities/cuda/cudnn_functions.h"
#include "cudnn_kernel.hh"
#include "runtime/mem_manager.hh"

namespace refactor::kernel {
    using namespace cudnn;
    using namespace runtime;

    auto ConvCudnn::lower(Resources &res) const noexcept -> Routine {
        // RAII for closure
        struct Descriptors {
            hipdnnTensorDescriptor_t x, y;
            hipdnnFilterDescriptor_t w;
            hipdnnConvolutionDescriptor_t conv;
            hipdnnConvolutionFwdAlgo_t algo;
            size_t workspaceSize;

            Descriptors() : workspaceSize(0) {
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&x));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&y));
                CUDNN_ASSERT(hipdnnCreateFilterDescriptor(&w));
                CUDNN_ASSERT(hipdnnCreateConvolutionDescriptor(&conv));
            }
            ~Descriptors() {
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(x));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(y));
                CUDNN_ASSERT(hipdnnDestroyFilterDescriptor(w));
                CUDNN_ASSERT(hipdnnDestroyConvolutionDescriptor(conv));
            }

            Descriptors(const Descriptors &) = delete;
            Descriptors(Descriptors &&) = delete;
        };
        auto d = std::make_shared<Descriptors>();

        auto cudnnDataType = cudnnDataTypeConvert(info.dt);
        auto xs = info.xShape, ys = info.yShape, ws = info.wShape;
        CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(d->x, HIPDNN_TENSOR_NCHW, cudnnDataType, xs[0], xs[1], xs[2], xs[3]));
        CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(d->y, HIPDNN_TENSOR_NCHW, cudnnDataType, ys[0], ys[1], ys[2], ys[3]));
        CUDNN_ASSERT(hipdnnSetFilter4dDescriptor(d->w, cudnnDataType, HIPDNN_TENSOR_NCHW, ws[0], ws[1], ws[2], ws[3]));
        auto pp = info.pad;
        auto ss = info.stride;
        auto dd = info.dilation;
        CUDNN_ASSERT(hipdnnSetConvolution2dDescriptor(d->conv, pp[0], pp[1], ss[0], ss[1], dd[0], dd[1], HIPDNN_CROSS_CORRELATION, cudnnDataType));

        auto handle = res.fetchOrStore<CudnnContext>()->handle;
        int returnedAlgoCount;
        hipdnnConvolutionFwdAlgoPerf_t perfResults;
        CUDNN_ASSERT(hipdnnFindConvolutionForwardAlgorithm(
            handle,
            d->x, d->w, d->conv, d->y,
            1, &returnedAlgoCount, &perfResults));
        ASSERT(returnedAlgoCount == 1, "returnedAlgoCount != 1");
        d->algo = perfResults.algo;
        CUDNN_ASSERT(hipdnnGetConvolutionForwardWorkspaceSize(
            handle,
            d->x, d->w, d->conv, d->y,
            perfResults.algo,
            &d->workspaceSize));
        // nvcc at c++11 doesn't support real move capture
        return [d_ = std::move(d)](Resources &res, void const **inputs, void **outputs) {
            using mem_manager::ForeignBlob;
            auto const &d = *d_;
            // fetch cudnn handle from resources
            auto handle = res.fetchOrStore<CudnnContext>()->handle;
            auto workspace = ForeignBlob::share(res.fetch<MemManager>()->manager, d.workspaceSize);
            // TODO? build alpha/beta for double
            float alpha = 1, beta = 0;
            CUDNN_ASSERT(hipdnnConvolutionForward(
                handle,
                &alpha,
                d.x, inputs[0],
                d.w, inputs[1],
                d.conv, d.algo,
                *workspace, d.workspaceSize,
                &beta,
                d.y, outputs[0]));
        };
    }

}// namespace refactor::kernel
