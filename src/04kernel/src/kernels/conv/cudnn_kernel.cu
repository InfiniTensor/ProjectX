#include "hip/hip_runtime.h"
﻿#include "../../utilities/cuda/cudnn_context.hh"
#include "../../utilities/cuda/cudnn_functions.h"
#include "../expand/cuda_kernel.hh"
#include "../extra_padding/extra_padding.cuh"
#include "cudnn_kernel.hh"
#include "hardware/functions.h"

namespace refactor::kernel {
    using namespace cudnn;
    using namespace runtime;

    auto ConvCudnn::lower(Resources &res) const -> RoutineWorkspace {
        // RAII for closure
        struct Descriptors {
            hipdnnTensorDescriptor_t x, y;
            hipdnnFilterDescriptor_t w;
            hipdnnConvolutionDescriptor_t conv;
            hipdnnConvolutionFwdAlgo_t algo;
            std::optional<ExtraPadding> extraPadding;
            std::optional<Routine> biasExpand;
            bool f64;

            Descriptors(bool f64_) : extraPadding(std::nullopt),
                                     biasExpand(std::nullopt),
                                     f64(f64_) {
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&x));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&y));
                CUDNN_ASSERT(hipdnnCreateFilterDescriptor(&w));
                CUDNN_ASSERT(hipdnnCreateConvolutionDescriptor(&conv));
            }
            ~Descriptors() noexcept(false) {
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(x));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(y));
                CUDNN_ASSERT(hipdnnDestroyFilterDescriptor(w));
                CUDNN_ASSERT(hipdnnDestroyConvolutionDescriptor(conv));
            }

            Descriptors(const Descriptors &) = delete;
            Descriptors(Descriptors &&) = delete;
        };
        auto d = std::make_shared<Descriptors>(info.dt == DataType::F64);
        d->extraPadding = ExtraPadding::build(info.dt, info.xShape, info.pad);
        if (info.biasExpand) {
            d->biasExpand = ExpandCuda(*info.biasExpand).lower(res).routine;
        }
        int xs[]{
            info.xShape[0],
            info.xShape[1],
            info.xShape[2] + std::abs(info.pad[0] - info.pad[2]),
            info.xShape[3] + std::abs(info.pad[1] - info.pad[3]),
        };

        auto cudnnDataType = cudnnDataTypeConvert(info.dt);
        setCudnnTensor(d->x, info.dt, slice(xs, 4));
        setCudnnTensor(d->y, info.dt, slice(info.yShape, 4));
        auto ws = info.wShape;
        CUDNN_ASSERT(hipdnnSetFilter4dDescriptor(d->w, cudnnDataType, HIPDNN_TENSOR_NCHW, ws[0], ws[1], ws[2], ws[3]));
        auto pp = info.pad;
        auto ss = info.stride;
        auto dd = info.dilation;
        CUDNN_ASSERT(hipdnnSetConvolution2dDescriptor(
            d->conv,
            std::min(pp[0], pp[2]), std::min(pp[1], pp[3]),
            ss[0], ss[1],
            dd[0], dd[1],
            HIPDNN_CROSS_CORRELATION,
            cudnnDataType));

        if (auto group = xs[1] / ws[1]; group > 1) {
            CUDNN_ASSERT(hipdnnSetConvolutionGroupCount(d->conv, group));
        }

        auto handle = res.fetchOrStore<CudnnContext>()->handle;
        {
            int returnedAlgoCount;
            hipdnnConvolutionFwdAlgoPerf_t perfResults;
            CUDNN_ASSERT(hipdnnFindConvolutionForwardAlgorithm(
                handle,
                d->x, d->w, d->conv, d->y,
                1, &returnedAlgoCount, &perfResults));
            ASSERT(returnedAlgoCount == 1, "returnedAlgoCount != 1");
            d->algo = perfResults.algo;
            // for high accuracy, use this algo only
            // d->algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
        }
        size_t workspaceSize;
        {
            CUDNN_ASSERT(hipdnnGetConvolutionForwardWorkspaceSize(
                handle,
                d->x, d->w, d->conv, d->y,
                d->algo,
                &workspaceSize));
        }
        if (d->extraPadding) {
            workspaceSize = hardware::alignBytes(workspaceSize, 256);
        }

        // nvcc at c++11 doesn't support real move capture
        auto routine = [d, workspaceSize](Resources &res, void *workspace, void const *const *inputs, void *const *outputs) {
            void const *x = inputs[0], *w = inputs[1];
            if (auto f = d->extraPadding; f) {
                x = (*f)(x, reinterpret_cast<uint8_t *>(workspace) + workspaceSize);
            }
            if (auto f = d->biasExpand; f) {
                (*f)(res, workspace, inputs + 2, outputs);
            }
            // build alpha/beta for double
            union {
                float f32[2];
                double f64[2];
            };
            void *alpha, *beta;
            if (d->f64) {
                f64[0] = 1;
                f64[1] = d->biasExpand ? 1 : 0;
                alpha = f64;
                beta = f64 + 1;
            } else {
                f32[0] = 1;
                f32[1] = d->biasExpand ? 1 : 0;
                alpha = f32;
                beta = f32 + 1;
            }
            CUDNN_ASSERT(hipdnnConvolutionForward(
                res.fetchOrStore<CudnnContext>()->handle,
                alpha,
                d->x, x,
                d->w, w,
                d->conv, d->algo,
                workspace, workspaceSize,
                beta,
                d->y, outputs[0]));
        };
        return {
            std::move(routine),
            workspaceSize + (d->extraPadding ? d->extraPadding->workspace() : 0),
        };
    }

}// namespace refactor::kernel
