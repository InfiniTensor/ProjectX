#include "hip/hip_runtime.h"
﻿#include "cuda_kernel.hh"
#include "kernel/cuda/split.cuh"
#include "mem_manager/foreign_blob.hh"
#include "runtime/mem_manager.hh"
#include <thrust/device_vector.h>

namespace refactor::kernel {
    using namespace runtime;

    Routine SplitCuda::lower(Resources &) const noexcept {
        auto sub = std::min(info.submultiple(), 16u);
        return [segments = thrust::device_vector<uint_lv2>(info.segments.begin(), info.segments.end()),
                params = cuda::ThreadsDistributer()(info.blockCount * info.sum / sub),
                sum = info.sum / sub,
                sub](Resources &res, void const **inputs, void **outputs) {
            auto size = segments.size() * sizeof(void *);
            auto outputs_ = mem_manager::ForeignBlob::share(res.fetch<MemManager>()->manager, size);
            outputs_->copyIn(outputs, size);
            cuda::launchSplit(
                params,
                inputs[0],
                segments.data().get(),
                reinterpret_cast<void **>((void *) *outputs_),
                segments.size(),
                sum,
                sub);
        };
    }

}// namespace refactor::kernel
