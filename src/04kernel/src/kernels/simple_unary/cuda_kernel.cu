#include "hip/hip_runtime.h"
﻿#include "cuda_kernel.hh"
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>

namespace refactor::kernel {
    using K = SimpleUnaryCuda;
    using Op = SimpleUnaryType;
    using DT = DataType;

    template<class T> struct AbsFunctor {
        __device__ T operator()(T x) const { return abs(x); }
    };
    template<class T> struct NegFunctor {
        __device__ T operator()(T x) const { return -x; }
    };
    template<class T> struct ReluFunctor {
        __device__ T operator()(T x) const { return x > 0 ? x : 0; }
    };
    template<class T> struct SqrtFunctor {
        __device__ T operator()(T x) const {
            using M = std::conditional_t<sizeof(T) <= 4, float, double>;
            return static_cast<T>(sqrt(static_cast<M>(x)));
        }
    };
    template<class T> struct SigmoidFunctor {
        __device__ T operator()(T x) const {
            using M = std::conditional_t<sizeof(T) <= 4, float, double>;
            return static_cast<T>(1 / (1 + std::exp(-static_cast<M>(x))));
        }
    };
    template<class T> struct TanhFunctor {
        __device__ T operator()(T x) const {
            using M = std::conditional_t<sizeof(T) <= 4, float, double>;
            return static_cast<T>(tanh(static_cast<M>(x)));
        }
    };

    template<class T, class UnaryFuntor>
    auto lowerTyped(size_t n) noexcept {
        using namespace runtime;

        return [n](Resources &, void const **inputs, void **outputs) {
            auto const *x = static_cast<T const *>(inputs[0]);
            auto *y = static_cast<T *>(outputs[0]);
            thrust::transform(thrust::device, x, x + n, y, UnaryFuntor{});
        };
    }
    template<class T>
    auto copyTyped(size_t n) noexcept {
        using namespace runtime;

        return [n](Resources &, void const **inputs, void **outputs) {
            auto const *x = static_cast<T const *>(inputs[0]);
            auto *y = static_cast<T *>(outputs[0]);
            thrust::copy_n(thrust::device, x, n, y);
        };
    }

#define CASE(FUNC, TYPE) \
    case DT::TYPE:       \
        return lowerTyped<primitive<DT::TYPE>::type, FUNC##Functor<primitive<DT::TYPE>::type>>(size)
#define COPY(TYPE) \
    case DT::TYPE: \
        return copyTyped<primitive<DT::TYPE>::type>(size)
#define GROUP_F(FUNC) \
    CASE(FUNC, F32);  \
    CASE(FUNC, F64)
#define GROUP_I(FUNC) \
    CASE(FUNC, I8);   \
    CASE(FUNC, I16);  \
    CASE(FUNC, I32);  \
    CASE(FUNC, I64)
#define GROUP_U(FUNC) \
    CASE(FUNC, U8);   \
    CASE(FUNC, U16);  \
    CASE(FUNC, U32);  \
    CASE(FUNC, U64)

    Routine K::lower(Resources &) const noexcept {
        switch (opType) {
            case Op::Abs:
                switch (dataType) {
                    GROUP_F(Abs);
                    GROUP_I(Abs);
                    COPY(U8);
                    COPY(U16);
                    COPY(U32);
                    COPY(U64);
                    default:
                        UNREACHABLE();
                }
            case Op::Relu:
                switch (dataType) {
                    GROUP_F(Relu);
                    GROUP_I(Relu);
                    COPY(U8);
                    COPY(U16);
                    COPY(U32);
                    COPY(U64);
                    default:
                        UNREACHABLE();
                }
            case Op::Sqrt:
                switch (dataType) {
                    GROUP_F(Sqrt);
                    GROUP_I(Sqrt);
                    GROUP_U(Sqrt);
                    default:
                        UNREACHABLE();
                }
            case Op::Sigmoid:
                switch (dataType) {
                    GROUP_F(Sigmoid);
                    GROUP_I(Sigmoid);
                    GROUP_U(Sigmoid);
                    default:
                        UNREACHABLE();
                }
            case Op::Tanh:
                switch (dataType) {
                    GROUP_F(Tanh);
                    GROUP_I(Tanh);
                    GROUP_U(Tanh);
                    default:
                        UNREACHABLE();
                }
            case Op::Neg:
                switch (dataType) {
                    GROUP_F(Neg);
                    GROUP_I(Neg);
                    default:
                        UNREACHABLE();
                }
            default:
                UNREACHABLE();
        }
    }

}// namespace refactor::kernel
