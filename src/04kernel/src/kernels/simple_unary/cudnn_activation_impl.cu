﻿#include "../../utilities/cuda/cudnn_context.hh"
#include "../../utilities/cuda/cudnn_functions.h"
#include "cudnn_activation_impl.hh"
#include <hipDNN.h>

namespace refactor::kernel::cudnn {
    using namespace runtime;
    using Op = SimpleUnaryType;

    Routine lower(Op op, common::DataType dt, int size) noexcept {
        // RAII for closure
        struct Descriptors {
            hipdnnActivationDescriptor_t activation;
            hipdnnTensorDescriptor_t tensor;

            Descriptors() : activation(nullptr), tensor(nullptr) {
                CUDNN_ASSERT(hipdnnCreateActivationDescriptor(&activation));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&tensor));
            }
            ~Descriptors() {
                CUDNN_ASSERT(hipdnnDestroyActivationDescriptor(activation));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(tensor));
            }

            Descriptors(const Descriptors &) = delete;
            Descriptors(Descriptors &&) = delete;
        };
        auto d = std::make_shared<Descriptors>();

        // clang-format off
        hipdnnActivationMode_t
        mode = op == Op::Relu ? HIPDNN_ACTIVATION_RELU
             : op == Op::Sigmoid ? HIPDNN_ACTIVATION_SIGMOID
             : op == Op::Tanh ? HIPDNN_ACTIVATION_TANH
             : UNREACHABLEX(hipdnnActivationMode_t, "");
        // clang-format on
        int stride = 1;

        CUDNN_ASSERT(hipdnnSetActivationDescriptor(d->activation, mode, HIPDNN_PROPAGATE_NAN, 0.0));
        CUDNN_ASSERT(hipdnnSetTensorNdDescriptor(d->tensor, cudnnDataTypeConvert(dt), 1, &size, &stride));

        // nvcc at c++11 doesn't support real move capture
        return [d = std::move(d)](Resources &res, Addresses inputs, Addresses outputs) {
            // fetch cudnn handle from resources
            auto handle = res.fetchOrStore<CudnnContext>()->handle;
            // name inputs and outputs
            auto x = inputs[0];
            auto y = outputs[0];
            // call cudnn activation
            float alpha = 1, beta = 0;
            CUDNN_ASSERT(hipdnnActivationForward(handle, d->activation, &alpha, d->tensor, x, &beta, d->tensor, y));
        };
    }

}// namespace refactor::kernel::cudnn
