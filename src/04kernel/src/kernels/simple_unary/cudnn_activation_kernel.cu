﻿#include "../../utilities/cuda/cudnn_context.hh"
#include "../../utilities/cuda/cudnn_functions.h"
#include "cudnn_activation_kernel.hh"
#include <hipDNN.h>

namespace refactor::kernel {
    using namespace cudnn;
    using namespace runtime;
    using Ty = SimpleUnaryType;

    Routine ActivationCudnn::lower(Resources &res) const noexcept {
        // RAII for closure
        struct Descriptors {
            hipdnnActivationDescriptor_t activation;
            hipdnnTensorDescriptor_t tensor;

            Descriptors() : activation(nullptr), tensor(nullptr) {
                CUDNN_ASSERT(hipdnnCreateActivationDescriptor(&activation));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&tensor));
            }
            ~Descriptors() {
                CUDNN_ASSERT(hipdnnDestroyActivationDescriptor(activation));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(tensor));
            }

            Descriptors(const Descriptors &) = delete;
            Descriptors(Descriptors &&) = delete;
        };
        auto d = std::make_shared<Descriptors>();

        // clang-format off
        hipdnnActivationMode_t
        mode = type == Ty::Relu    ? HIPDNN_ACTIVATION_RELU
             : type == Ty::Sigmoid ? HIPDNN_ACTIVATION_SIGMOID
             : type == Ty::Tanh    ? HIPDNN_ACTIVATION_TANH
             : UNREACHABLEX(hipdnnActivationMode_t, "");
        // clang-format on

        CUDNN_ASSERT(hipdnnSetActivationDescriptor(d->activation, mode, HIPDNN_PROPAGATE_NAN, 0.0));
        CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(d->tensor, HIPDNN_TENSOR_NCHW, cudnnDataTypeConvert(dataType), 1, 1, 1, size));

        res.fetchOrStore<CudnnContext>();
        // nvcc at c++11 doesn't support real move capture
        return [d = std::move(d)](Resources &res, void *workspace, void const *const *inputs, void *const *outputs) {
            // fetch cudnn handle from resources
            auto handle = res.fetchOrStore<CudnnContext>()->handle;
            // name inputs and outputs
            auto x = inputs[0];
            auto y = outputs[0];
            // call cudnn activation
            float alpha = 1, beta = 0;
            CUDNN_ASSERT(hipdnnActivationForward(handle, d->activation, &alpha, d->tensor, x, &beta, d->tensor, y));
        };
    }

}// namespace refactor::kernel
