#include "../../utilities/cuda/cudnn_context.hh"
#include "../../utilities/cuda/cudnn_functions.h"
#include "common.h"
#include "cudnn_kernel.hh"
#include "runtime/mem_manager.hh"

namespace refactor::kernel {
    using namespace cudnn;
    using namespace runtime;

    Routine ReduceCudnn::lower(Resources &res) const noexcept {
        // RAII for closure
        struct Descriptors {
            hipdnnTensorDescriptor_t inDesc;
            hipdnnTensorDescriptor_t outDesc;

            hipdnnReduceTensorDescriptor_t reduceDesc;
            size_t workspaceSize;
            size_t idxWorkspaceSize;

            Descriptors() : workspaceSize(0), idxWorkspaceSize(0) {
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&inDesc));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&outDesc));
                CUDNN_ASSERT(hipdnnCreateReduceTensorDescriptor(&reduceDesc));
            }
            ~Descriptors() {
                // Destories in CUDA does not require sync. But cuDNN does not state
                // whether sync is required before destories.
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(inDesc));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(outDesc));
                CUDNN_ASSERT(hipdnnDestroyReduceTensorDescriptor(reduceDesc));
            }

            Descriptors(const Descriptors &) = delete;
            Descriptors(Descriptors &&) = delete;
        };
        auto d = std::make_shared<Descriptors>();
        auto handler = res.fetchOrStore<CudnnContext>()->handle;

        // Each dimension of the output tensor C must match the corresponding
        // dimension of the input tensor A or must be equal to 1. The dimensions
        // equal to 1 indicate the dimensions of A to be reduced.
        int nInDims = shape.size();
        std::vector<int> inDimArray, outDimArray, inStrideArray, outStrideArray;
        size_t stride = 1;
        for (int i = nInDims - 1; i >= 0; --i) {
            inDimArray.insert(inDimArray.begin(), shape[i]);
            inStrideArray.insert(inStrideArray.begin(), stride);
            stride *= shape[i];
        }
        std::unordered_set axesSet(axes.begin(), axes.end());
        for (size_t i = 0; i < shape.size(); ++i) {
            if (axesSet.find(i) == axesSet.end()) {
                outDimArray.push_back(shape[i]);
            } else {
                outDimArray.push_back(1);
            }
        }
        size_t nOutDims = outDimArray.size();
        stride = 1;
        for (int i = nOutDims - 1; i >= 0; --i) {
            outStrideArray.insert(outStrideArray.begin(), stride);
            stride *= outDimArray[i];
        }

        // hipdnnSetTensorNdDescriptor is used when nDim>3, otherwise,it is
        // recomended to use hipdnnSetTensor4dDescriptor and set the unused
        // dimension size to 1.
        // get inputs outputs
        auto cudnnDataType = cudnnDataTypeConvert(dataType);
        if (nInDims > 3) {
            CUDNN_ASSERT(hipdnnSetTensorNdDescriptor(
                d->inDesc, cudnnDataType, nInDims, inDimArray.data(), inStrideArray.data()));
            CUDNN_ASSERT(hipdnnSetTensorNdDescriptor(
                d->outDesc, cudnnDataType, nOutDims, outDimArray.data(), outStrideArray.data()));
        } else {
            int idims[4] = {1, 1, 1, 1}, odims[4] = {1, 1, 1, 1};
            for (int i = 0; i < nInDims; ++i) {
                idims[4 - i - 1] = inDimArray[nInDims - i - 1];
            }
            for (int i = 0; i < nOutDims; ++i) {
                odims[4 - i - 1] = outDimArray[nOutDims - i - 1];
            }

            CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(
                d->inDesc, HIPDNN_TENSOR_NCHW, cudnnDataType, idims[0], idims[1],
                idims[2], idims[3]));
            CUDNN_ASSERT(hipdnnSetTensor4dDescriptor(
                d->outDesc, HIPDNN_TENSOR_NCHW, cudnnDataType, odims[0],
                odims[1], odims[2], odims[3]));
        }

        // get reduce descriptor
        hipdnnReduceTensorOp_t reduceOp = HIPDNN_REDUCE_TENSOR_ADD;
        switch (reduceType) {
            case ReduceType::Mean:
                reduceOp = HIPDNN_REDUCE_TENSOR_AVG;
                break;
            case ReduceType::Min:
                reduceOp = HIPDNN_REDUCE_TENSOR_MIN;
                break;
            case ReduceType::Max:
                reduceOp = HIPDNN_REDUCE_TENSOR_MAX;
                break;
            case ReduceType::L1:
                reduceOp = HIPDNN_REDUCE_TENSOR_NORM1;
                break;
            case ReduceType::L2:
                reduceOp = HIPDNN_REDUCE_TENSOR_NORM2;
                break;
            case ReduceType::Sum:
                reduceOp = HIPDNN_REDUCE_TENSOR_ADD;
                break;
            case ReduceType::Prod:
                reduceOp = HIPDNN_REDUCE_TENSOR_MUL;
                break;
            default:
                UNREACHABLE();
        };
        CUDNN_ASSERT(hipdnnSetReduceTensorDescriptor(
            d->reduceDesc, reduceOp, cudnnDataType,
            HIPDNN_NOT_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES,
            HIPDNN_32BIT_INDICES));

        // get workspace
        CUDNN_ASSERT(
            hipdnnGetReductionWorkspaceSize(handler, d->reduceDesc,
                                           d->inDesc, d->outDesc, &d->workspaceSize));

        // get index workspace
        CUDNN_ASSERT(
            cudnnGetReductionIndicesSize(handler, d->reduceDesc,
                                         d->inDesc, d->outDesc, &d->idxWorkspaceSize));


        // nvcc at c++11 doesn't support real move capture
        return [d_ = std::move(d)](Resources &res, void const **inputs, void **outputs) {
            using mem_manager::ForeignBlob;
            // fetch cudnn handle from resources
            auto handle = res.fetchOrStore<CudnnContext>()->handle;
            auto const &d = *d_;
            auto wsData = ForeignBlob::share(res.fetch<runtime::MemManager>()->manager, d.workspaceSize);
            auto idxWsData = ForeignBlob::share(res.fetch<runtime::MemManager>()->manager, d.idxWorkspaceSize);

            // name inputs and outputs
            auto inData = inputs[0];
            auto outData = outputs[0];
            // reduce
            float alpha = 1.f, beta = 0.f;
            CUDNN_ASSERT(hipdnnReduceTensor(handle, d.reduceDesc,
                                           *idxWsData, d.idxWorkspaceSize, *wsData,
                                           d.workspaceSize, &alpha, d.inDesc, inData,
                                           &beta, d.outDesc, outData));
        };
    }

}// namespace refactor::kernel
