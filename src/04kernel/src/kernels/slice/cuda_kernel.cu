﻿#include "cuda_kernel.hh"
#include "kernel/cuda/slice.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace refactor::kernel {
    using namespace runtime;

    Routine SliceCuda::lower(Resources &) const noexcept {
        auto reformed = info.reform(16);
        thrust::host_vector<cuda::DimInfo> dims(info.dims.size());
        std::transform(info.dims.begin(), info.dims.end(),
                       dims.begin(),
                       [](auto const &d) { return cuda::DimInfo{
                                               d.countStride,
                                               d.sizeStart,
                                               d.sizeStride,
                                           }; });
        return [dims = thrust::device_vector<cuda::DimInfo>(dims),
                params = cuda::ThreadsDistributer()(reformed.blockCount),
                blockSize = reformed.blockSize,
                baseOffset = reformed.baseOffset](Resources &, void const **inputs, void **outputs) {
            auto src = reinterpret_cast<uint8_t const *>(inputs[0]) + baseOffset;
            cuda::launchSlice(params, src, dims.data().get(), outputs[0],
                              dims.size(),
                              blockSize);
        };
    }

}// namespace refactor::kernel
