﻿#include "cuda_kernel.hh"
#include "kernel/cuda/slice.cuh"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace refactor::kernel {
    using namespace runtime;

    auto SliceCuda::lower(Resources &) const noexcept -> RoutineWorkspace {
        thrust::host_vector<cuda::DimInfo> dims(info.dims.size());
        std::transform(info.dims.begin(), info.dims.end(),
                       dims.begin(),
                       [](auto const &d) {
                           return cuda::DimInfo{
                               d.strideO,
                               d.skip,
                               d.strideI,
                           };
                       });
        return [dims = thrust::device_vector<cuda::DimInfo>(dims),
                params = cuda::ThreadsDistributer()(info.blockCount),
                blockSize = info.blockSize](Resources &, void *workspace, void const *const *inputs, void *const *outputs) {
            auto src = reinterpret_cast<uint8_t const *>(inputs[0]);
            cuda::launchSlice(params, src, dims.data().get(), outputs[0],
                              dims.size(),
                              blockSize);
        };
    }

}// namespace refactor::kernel
