﻿#include "common/error_handler.h"
#include "cudnn_context.hh"
#include <hipDNN.h>

namespace refactor::kernel::cudnn {

    CudnnContext::CudnnContext() noexcept
        : runtime::Resource() {
        hipdnnHandle_t handle_;
        if (hipdnnCreate(&handle_) != HIPDNN_STATUS_SUCCESS) {
            RUNTIME_ERROR("Failed to create cudnn handle");
        }
        handle = handle_;
    }
    CudnnContext::~CudnnContext() noexcept {
        auto handle_ = std::any_cast<hipdnnHandle_t>(handle);
        if (hipdnnDestroy(handle_) != HIPDNN_STATUS_SUCCESS) {
            RUNTIME_ERROR("Failed to destroy cudnn handle");
        }
    }

    auto CudnnContext::typeId() noexcept -> size_t {
        static uint8_t ID = 1;
        return reinterpret_cast<size_t>(&ID);
    }
    auto CudnnContext::build() noexcept -> runtime::ResourceBox {
        return std::make_unique<CudnnContext>();
    }

    auto CudnnContext::resourceTypeId() const noexcept -> size_t {
        return typeId();
    }
    auto CudnnContext::description() const noexcept -> std::string_view {
        return "CudnnContext";
    }

}// namespace refactor::kernel::cudnn
