﻿#include "../cudnn_context.hh"
#include "../cudnn_error_handler.h"
#include "cudnn_impl.h"
#include "runtime/resource.h"
#include <hipDNN.h>
#include <utility>

namespace refactor::kernel::cudnn {
    using namespace runtime;
    using Ctx = CudnnContext;
    using DT = common::DataType;

    Operation Info::lower() const {
        int strideAx[4]{0, 0, 0, 1},       // to calculate
            dimAp[4]{1, dimAx[1], 1, 1},   // 1xCx1x1
            strideAp[4]{dimAx[1], 1, 1, 1};// Cx1x1x1
        // TODO: calculate real stride based on layout type
        for (auto i = 3; i > 0; --i) {
            strideAx[i - 1] = strideAx[i] * dimAx[i];
        }

        // RAII for closure
        struct Descriptors {
            hipdnnTensorDescriptor_t x, param;

            Descriptors() : x(nullptr), param(nullptr) {
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&x),
                             "hipdnnCreateTensorDescriptor failed");
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&param),
                             "hipdnnCreateTensorDescriptor failed");
            }
            ~Descriptors() {
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(x),
                             "hipdnnDestroyTensorDescriptor failed");
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(param),
                             "hipdnnDestroyTensorDescriptor failed");
            }

            Descriptors(const Descriptors &) = delete;
            Descriptors(Descriptors &&) = delete;
        };
        auto d = std::make_shared<Descriptors>();
        // clang-format off
        hipdnnDataType_t
            dtX_ = dtX == DT::F32  ? HIPDNN_DATA_FLOAT
                 : dtX == DT::F64  ? HIPDNN_DATA_DOUBLE
                 : dtX == DT::FP16 ? HIPDNN_DATA_HALF
                 : dtX == DT::BF16 ? CUDNN_DATA_BFLOAT16
                 : UNREACHABLEX(hipdnnDataType_t, ""),
            dtParam_ = dtParam == DT::F32 ? HIPDNN_DATA_FLOAT
                     : dtParam == DT::F64 ? HIPDNN_DATA_DOUBLE
                     : UNREACHABLEX(hipdnnDataType_t, "");
        // clang-format on

        CUDNN_ASSERT(hipdnnSetTensorNdDescriptor(d->x, dtX_, 4, dimAx, strideAx),
                     "hipdnnSetTensorNdDescriptor failed");
        CUDNN_ASSERT(hipdnnSetTensorNdDescriptor(d->param, dtParam_, 4, dimAp, strideAp),
                     "hipdnnSetTensorNdDescriptor failed");

        // nvcc at c++11 doesn't support real move capture
        return [d = std::move(d),
                param64 = dtParam == DT::F64,
                epsilon = this->epsilon](Resources &res, Addresses inputs, Addresses outputs) {
            // fetch cudnn handle from resources
            auto handle = std::any_cast<hipdnnHandle_t>(res.fetchOrStore<CudnnContext>()->handle);
            // name inputs and outputs
            auto x = inputs[0],
                 scale = inputs[1],
                 bias = inputs[2],
                 mean = inputs[3],
                 var = inputs[4];
            auto y = outputs[0];
            // build alpha/beta for double
            union {
                float f32[2];
                double f64[2];
            };
            void *alpha, *beta;
            if (param64) {
                f32[0] = 1;
                f32[1] = 0;
                alpha = f32;
                beta = f32 + 1;
            } else {
                f64[0] = 1;
                f64[1] = 0;
                alpha = f64;
                beta = f64 + 1;
            }
            CUDNN_ASSERT(hipdnnBatchNormalizationForwardInference(
                             handle, HIPDNN_BATCHNORM_SPATIAL, alpha, beta,
                             d->x, x,
                             d->x, y, // desc(x) === desc(y) for onnx
                             d->param,// scale, bias, mean, var
                             scale, bias, mean, var,
                             epsilon),
                         "hipdnnBatchNormalizationForwardInference failed");
        };
    }

}// namespace refactor::kernel::cudnn
