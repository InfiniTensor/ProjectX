﻿#include "../cudnn_context.hh"
#include "../cudnn_error_handler.h"
#include "cudnn_impl.h"
#include "runtime/resource.h"
#include <hipDNN.h>

namespace refactor::kernel::cudnn {
    using namespace runtime;
    using Ctx = CudnnContext;

    Operation lower(
        float epsilon,
        common::DataType dataType,
        Shape shape,
        uint32_t valueSize) {

        auto rank = shape.size();

        std::vector<int> ints(rank * 4, 1);
        auto dimAx = ints.data();
        auto strideAx = dimAx + rank;
        auto dimAp = strideAx + rank;
        auto strideAp = dimAp + rank;

        auto temp = 1;
        for (size_t i = 0; i < rank; ++i) {
            auto i_ = rank - i - 1;// reverse
            dimAx[i] = shape[i];
            strideAx[i_] = temp;
            temp *= shape[i];
        }
        dimAp[1] = valueSize;
        strideAp[0] = valueSize;

        hipdnnTensorDescriptor_t xDesc;
        CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&xDesc),
                     "hipdnnCreateTensorDescriptor failed");
        CUDNN_ASSERT(hipdnnSetTensorNdDescriptor(xDesc, HIPDNN_DATA_FLOAT, rank, dimAx, strideAx),
                     "hipdnnSetTensorNdDescriptor failed");

        hipdnnTensorDescriptor_t paraDesc;
        CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&paraDesc),
                     "hipdnnCreateTensorDescriptor failed");
        CUDNN_ASSERT(hipdnnSetTensorNdDescriptor(paraDesc, HIPDNN_DATA_FLOAT, rank, dimAp, strideAp),
                     "hipdnnSetTensorNdDescriptor failed");

        return [xDesc, paraDesc, epsilon](Resources &res, Addresses inputs, Addresses outputs) {
            auto handle = std::any_cast<hipdnnHandle_t>(res.fetchOrStore<CudnnContext>()->handle);
            auto x = inputs[0],
                 scale = inputs[1],
                 bias = inputs[2],
                 mean = inputs[3],
                 var = inputs[4];
            auto y = outputs[0];

            float alpha = 1, beta = 0;
            // This mode is intended for use after convolutional layers
            auto stat = hipdnnBatchNormalizationForwardInference(
                handle, HIPDNN_BATCHNORM_SPATIAL, &alpha, &beta,
                xDesc, x, xDesc, y, paraDesc, scale, bias, mean, var, epsilon);
            if (stat != HIPDNN_STATUS_SUCCESS) {
                RUNTIME_ERROR("hipdnnBatchNormalizationForwardInference failed");
            }
        };
    }

}// namespace refactor::kernel::cudnn
