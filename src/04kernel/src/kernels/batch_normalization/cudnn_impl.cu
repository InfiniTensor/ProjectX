﻿#include "../cudnn_context.hh"
#include "cudnn_impl.h"
#include "runtime/resource.h"
#include <hipDNN.h>

namespace refactor::kernel::cudnn {
    using namespace runtime;
    using Ctx = CudnnContext;

    Operation lower(
        float epsilon,
        common::DataType dataType,
        Shape shape,
        uint32_t valueSize) {
        return [](Resources &res, Addresses inputs, Addresses outputs) {
            auto handle = std::any_cast<hipdnnHandle_t>(res.fetchOrStore<CudnnContext>()->handle());
        };
    }

}// namespace refactor::kernel::cudnn
