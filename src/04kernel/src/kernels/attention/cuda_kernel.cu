#include "hip/hip_runtime.h"
﻿#include "../../utilities/cuda/cublaslt_utils.cuh"
#include "cuda_kernel.hh"
#include "hardware/functions.h"
#include "kernel/cuda/reduce.cuh"

namespace refactor::kernel {
    using K = AttentionCuda;
    using namespace hipcublas;

    // 因果系统的注意力遮罩。
    // tokenId: 第几个词
    //  seqLen: 此次处理的词数
    //   posId: 在 kv cache 中的位置
    //  attLen = pastSeqLen + seqLen
    static __forceinline__ __device__ bool
    causualMask(int tokenId, int seqLen,
                int posId, int attLen) {
        // tokenId ↓ |<---attLen---->|
        //         0 | * * ... *     |
        //         1 | * * ... * *   |
        //         2 | * * ... * * * |
        // seqLen: 3 |---------------|
        return attLen + tokenId >= posId + seqLen;
    }

    // gridDim.x = batch * nHead
    // gridDim.y = seqLen
    // blockDim.x = min(1024, attLen)
    template<class T>
    static __global__ void softmax(
        T *__restrict__ att,
        bool (*mask)(int, int, int, int),
        uint32_t attLen,
        uint32_t bufLen) {
        // 找到这个线程块对应的 attention 区域
        att += (blockIdx.x * gridDim.x + gridDim.y) * bufLen;
        // 将输入装入共享内存并 cast + mask
        extern __shared__ float shared[];// size = attLen = pastSeqLen + seqLen
        for (auto i = threadIdx.x; i < attLen; i += blockDim.x) {
            shared[i] = mask(blockIdx.y, gridDim.y, i, attLen) ? float(att[i]) : -__FLT_MAX__;
        }

        float localMax = -1e20;
        for (auto i = threadIdx.x; i < attLen; i += blockDim.x) {
            localMax = hipcub::Max()(localMax, shared[i]);
        }
        localMax = cuda::blockReduce(localMax, -1e20f, hipcub::Max());

        float localSum = 1e-20;
        for (auto i = threadIdx.x; i < attLen; i += blockDim.x) {
            localSum += shared[i] = expf(shared[i] - localMax);
        }
        localSum = cuda::blockReduce(localSum, 1e-20f, hipcub::Sum());
        auto reciprocal = fdividef(1, localSum);
        for (auto i = threadIdx.x; i < attLen; i += blockDim.x) {
            att[i] = shared[i] * reciprocal;
        }
    }

    RoutineWorkspace K::lower(Resources &res) const {
        auto handle = res.fetchOrStore<CublasLtContext>()->handle;

        constexpr auto ROW_MAJOR = HIPBLASLT_ORDER_ROW;
        constexpr auto COL_MAJOR = HIPBLASLT_ORDER_COL;

        if (!info.cacheLen) {
            if (info.nHead == info.nKVHead) {
                // RAII for closure
                struct Descriptors {
                    MatMulDescriptor mul;
                    MatrixDescriptor q, k, v, att;
                    hipblasLtMatmulAlgo_t algoQK, algoAV;
                    size_t attSize, workspaceSizeQK, workspaceSizeAV;

                    Descriptors(CublasLtContext const &context,
                                AttentionInfo info)
                        : mul(computeTypeConvert(info.dataType),
                              dataTypeConvert(info.dataType)),
                          q(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.headDim),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          k(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.headDim),
                              .cols = static_cast<uint64_t>(info.seqLen),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = COL_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          v(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.headDim),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          att(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.seqLen),
                              .majorStride = static_cast<int64_t>(info.seqLen),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.seqLen),
                          }),
                          attSize(info.batch * info.nHead * info.seqLen * info.seqLen * info.dataType.size()) {
                        auto [algoQK_, workspaceSizeQK_] = tune(context.handle, mul, q, k, att);
                        auto [algoAV_, workspaceSizeAV_] = tune(context.handle, mul, att, v, q);
                        algoQK = algoQK_;
                        algoAV = algoAV_;
                        workspaceSizeQK = workspaceSizeQK_;
                        workspaceSizeAV = workspaceSizeAV_;
                    }
                };

                auto const &context = *res.fetchOrStore<CublasLtContext>();
                auto d = std::make_shared<Descriptors>(context, info);
                auto workspaceSize = d->attSize;
                workspaceSize = hardware::alignBytes(workspaceSize, 256);
                workspaceSize += d->workspaceSizeQK;
                workspaceSize += d->workspaceSizeAV;
                workspaceSize = hardware::alignBytes(workspaceSize, 256);

                auto routine = [d = std::move(d), info = this->info]//
                    (Resources & res, void *workspace, void const *const *inputs, void *const *outputs) {
                        auto handle = res.fetchOrStore<CublasLtContext>()->handle;
                        auto q = inputs[0];
                        auto k = inputs[1];
                        auto v = inputs[2];
                        auto o = outputs[0];
                        auto att = reinterpret_cast<half *>(workspace);
                        auto workspaceQK = reinterpret_cast<uint8_t *>(workspace) + hardware::alignBytes(d->attSize, 256);
                        auto workspaceAV = workspaceQK + hardware::alignBytes(d->workspaceSizeQK, 256);
                        {
                            half alpha = rsqrtf(info.headDim), beta = 0;
                            hipblasLtMatmul(
                                handle, d->mul.get(),
                                &alpha,
                                q, d->q.get(),
                                k, d->k.get(),
                                &beta,
                                att, d->att.get(),
                                att, d->att.get(),
                                &d->algoQK,
                                workspaceQK, d->workspaceSizeQK,
                                hipStreamLegacy);
                        }
                        softmax<<<dim3(info.batch * info.nHead, info.seqLen), info.seqLen>>>(
                            att, causualMask, info.seqLen, info.seqLen);
                        {
                            half alpha = 1, beta = 0;
                            hipblasLtMatmul(
                                handle, d->mul.get(),
                                &alpha,
                                att, d->att.get(),
                                v, d->v.get(),
                                &beta,
                                o, d->q.get(),
                                o, d->q.get(),
                                &d->algoAV,
                                workspaceAV, d->workspaceSizeAV,
                                hipStreamLegacy);
                        };
                    };

                return {std::move(routine), workspaceSize};
            }
        }
        TODO("");
    }

}// namespace refactor::kernel
