#include "hip/hip_runtime.h"
﻿#include "../../utilities/cuda/cublaslt_utils.cuh"
#include "cuda_kernel.hh"
#include "hardware/functions.h"
#include "kernel/cuda/functions.cuh"
#include <cub/block/block_reduce.cuh>

namespace refactor::kernel {
    using K = AttentionCuda;
    using namespace hipcublas;

    // 因果系统的注意力遮罩。
    // tokenId: 第几个词
    //  seqLen: 此次处理的词数
    //   posId: 在 kv cache 中的位置
    //  attLen = pastSeqLen + seqLen
    struct AttentionCausualMask {
        __forceinline__ __device__ bool
        operator()(int tokenId, int seqLen,
                   int posId, int attLen) {
            // tokenId ↓ |<---attLen---->|
            //         0 | * * ... *     |
            //         1 | * * ... * *   |
            //         2 | * * ... * * * |
            // seqLen: 3 |---------------|
            return attLen + tokenId >= posId + seqLen;
        }
    };

    // gridDim.x = batch * nHead
    // gridDim.y = seqLen
    // blockDim.x = 1024
    // sizeof(shared) = attLen * sizeof(float)
    template<class T, class Mask>
    static __global__ void softmax(
        T *__restrict__ att,
        Mask mask,
        uint32_t attLen,
        uint32_t bufLen) {
        // 找到这个线程块对应的 attention 区域
        att += (blockIdx.x * gridDim.y + blockIdx.y) * bufLen;
        // 将输入装入共享内存并 cast + mask
        extern __shared__ float shared[];// size = attLen = pastSeqLen + seqLen
        for (auto i = threadIdx.x; i < attLen; i += blockDim.x) {
            shared[i] = mask(blockIdx.y, gridDim.y, i, attLen) ? float(att[i]) : -__FLT_MAX__;
        }

        using BlockReduce = hipcub::BlockReduce<float, 1024>;
        __shared__ typename BlockReduce::TempStorage tempStorage;
        __shared__ float sharedMax, sharedSum;

        float localMax = -1e20;
        for (auto i = threadIdx.x; i < attLen; i += blockDim.x) {
            localMax = hipcub::Max()(localMax, shared[i]);
        }
        localMax = BlockReduce(tempStorage).Reduce(localMax, hipcub::Max(), attLen);
        if (threadIdx.x == 0) { sharedMax = localMax; }
        __syncthreads();

        float localSum = 1e-20;
        for (auto i = threadIdx.x; i < attLen; i += blockDim.x) {
            localSum += shared[i] = expf(shared[i] - sharedMax);
        }
        localSum = BlockReduce(tempStorage).Reduce(localSum, hipcub::Sum(), attLen);
        if (threadIdx.x == 0) { sharedSum = localSum; }
        __syncthreads();

        auto reciprocal = fdividef(1, sharedSum);
        for (auto i = threadIdx.x; i < attLen; i += blockDim.x) {
            att[i] = shared[i] * reciprocal;
        }
    }

    static __global__ void concatCache(
        void *__restrict__ cache,
        void const *__restrict__ value,
        dim_t pageStrideI,
        dim_t pageStrideO,
        dim_t pastOffset,
        dim_t n_items) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        if (tid < n_items) {
            auto dst = tid / pageStrideI * pageStrideO + pastOffset + (tid % pageStrideI);
            reinterpret_cast<float4 *>(cache)[dst] = reinterpret_cast<float4 const *>(value)[tid];
        }
    }
    constexpr uint64_t DYNAMIC_WORKSPACE_SIZE = 40 << 20;// 试出来 40MiB 是够用的

    RoutineWorkspace K::lower(Resources &res) const {
        auto handle = res.fetchOrStore<CublasLtContext>()->handle;

        constexpr auto ROW_MAJOR = HIPBLASLT_ORDER_ROW;
        constexpr auto COL_MAJOR = HIPBLASLT_ORDER_COL;

        if (!info.cacheLen) {
            if (info.nHead == info.nKVHead) {
                // RAII for closure
                struct Descriptors {
                    MatMulDescriptor mul;
                    MatrixDescriptor q, k, v, att;
                    hipblasLtMatmulAlgo_t algoQK, algoAV;
                    size_t workspaceSizeQK, workspaceSizeAV;

                    Descriptors(CublasLtContext const &context,
                                AttentionInfo info)
                        : mul(computeTypeConvert(info.dataType),
                              dataTypeConvert(info.dataType)),
                          q(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.headDim),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          k(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.headDim),
                              .cols = static_cast<uint64_t>(info.seqLen),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = COL_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          v(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.headDim),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          att(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.seqLen),
                              .majorStride = static_cast<int64_t>(info.seqLen),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.seqLen),
                          }) {
                        auto [algoQK_, workspaceSizeQK_] = tune(context.handle, mul, q, k, att, DYNAMIC_WORKSPACE_SIZE);
                        auto [algoAV_, workspaceSizeAV_] = tune(context.handle, mul, att, v, q, DYNAMIC_WORKSPACE_SIZE);
                        algoQK = algoQK_;
                        algoAV = algoAV_;
                        workspaceSizeQK = workspaceSizeQK_;
                        workspaceSizeAV = workspaceSizeAV_;
                    }
                };

                auto const &context = *res.fetchOrStore<CublasLtContext>();
                auto d = std::make_shared<Descriptors>(context, info);
                auto workspaceSize = info.attSize(0);
                workspaceSize = hardware::alignBytes(workspaceSize, 256);
                workspaceSize += d->workspaceSizeQK;
                workspaceSize += d->workspaceSizeAV;
                workspaceSize = hardware::alignBytes(workspaceSize, 256);

                auto routine = [d = std::move(d), info = this->info]//
                    (Resources & res, void *workspace, void const *const *inputs, void *const *outputs) {
                        auto handle = res.fetchOrStore<CublasLtContext>()->handle;
                        auto q = inputs[0];
                        auto k = inputs[1];
                        auto v = inputs[2];
                        auto o = outputs[0];
                        auto att = reinterpret_cast<half *>(workspace);
                        auto workspaceQK = reinterpret_cast<uint8_t *>(workspace) + hardware::alignBytes(info.attSize(0), 256);
                        auto workspaceAV = workspaceQK + hardware::alignBytes(d->workspaceSizeQK, 256);
                        auto stream = hipStreamLegacy;
                        {
                            half alpha = rsqrtf(info.headDim), beta = 0;
                            hipblasLtMatmul(
                                handle, d->mul.get(),
                                &alpha,
                                q, d->q.get(),
                                k, d->k.get(),
                                &beta,
                                att, d->att.get(),
                                att, d->att.get(),
                                &d->algoQK,
                                workspaceQK, d->workspaceSizeQK,
                                stream);
                        }
                        auto attLen = info.attLen(0);
                        auto bufLen = attLen;
                        softmax<<<dim3(info.batch * info.nHead, info.seqLen),
                                  std::min(1024u, attLen),
                                  attLen * sizeof(float),
                                  stream>>>(
                            att, AttentionCausualMask(), attLen, bufLen);
                        {
                            half alpha = 1, beta = 0;
                            hipblasLtMatmul(
                                handle, d->mul.get(),
                                &alpha,
                                att, d->att.get(),
                                v, d->v.get(),
                                &beta,
                                o, d->q.get(),
                                o, d->q.get(),
                                &d->algoAV,
                                workspaceAV, d->workspaceSizeAV,
                                stream);
                        }
                    };

                return {std::move(routine), workspaceSize};
            }
            TODO("");
        }
        if (info.concatCache && !info.resetCache) {
            if (info.nHead == info.nKVHead) {

                // RAII for closure
                struct Descriptors {
                    MatMulDescriptor mul;

                    Descriptors(AttentionInfo info)
                        : mul(computeTypeConvert(info.dataType),
                              dataTypeConvert(info.dataType)) {}
                };

                auto const &context = *res.fetchOrStore<CublasLtContext>();
                auto d = std::make_shared<Descriptors>(info);
                auto attentionSize = info.maxAttSize();
                auto workspaceSize = DYNAMIC_WORKSPACE_SIZE + attentionSize;

                for (auto attLen = 0; attLen < 2048; ++attLen) {
                    MatrixDescriptor
                        q_(MatrixLayout{
                            .dataType = dataTypeConvert(info.dataType),
                            .rows = static_cast<uint64_t>(info.seqLen),
                            .cols = static_cast<uint64_t>(info.headDim),
                            .majorStride = static_cast<int64_t>(info.headDim),
                            .order = ROW_MAJOR,
                            .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                            .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                        }),
                        k_(MatrixLayout{
                            .dataType = dataTypeConvert(info.dataType),
                            .rows = static_cast<uint64_t>(info.headDim),
                            .cols = static_cast<uint64_t>(attLen),
                            .majorStride = static_cast<int64_t>(info.headDim),
                            .order = COL_MAJOR,
                            .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                            .batchStride = static_cast<int64_t>(info.cacheLen * info.headDim),
                        }),
                        v_(MatrixLayout{
                            .dataType = dataTypeConvert(info.dataType),
                            .rows = static_cast<uint64_t>(attLen),
                            .cols = static_cast<uint64_t>(info.headDim),
                            .majorStride = static_cast<int64_t>(info.headDim),
                            .order = ROW_MAJOR,
                            .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                            .batchStride = static_cast<int64_t>(info.cacheLen * info.headDim),
                        }),
                        att_(MatrixLayout{
                            .dataType = dataTypeConvert(info.dataType),
                            .rows = static_cast<uint64_t>(info.seqLen),
                            .cols = static_cast<uint64_t>(attLen),
                            .majorStride = static_cast<int64_t>(info.cacheLen),
                            .order = ROW_MAJOR,
                            .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                            .batchStride = static_cast<int64_t>(info.cacheLen * info.seqLen),
                        });
                    tune(handle, d->mul,
                         q_, k_, att_,
                         DYNAMIC_WORKSPACE_SIZE);
                    tune(handle, d->mul,
                         att_, v_, q_,
                         DYNAMIC_WORKSPACE_SIZE);
                }

                auto routine = [d = std::move(d), info = this->info]//
                    (Resources & res, void *workspace, void const *const *inputs, void *const *outputs) {
                        auto handle = res.fetchOrStore<CublasLtContext>()->handle;
                        auto q = inputs[0];
                        auto k = inputs[1];
                        auto v = inputs[2];
                        int64_t past;
                        hipMemcpy(&past, inputs[3], sizeof(int64_t), hipMemcpyDeviceToHost);
                        auto attLen = info.attLen(past);
                        auto o = reinterpret_cast<half *>(outputs[0]);
                        auto kCache = reinterpret_cast<half *>(outputs[1]);
                        auto vCache = reinterpret_cast<half *>(outputs[2]);
                        auto att = reinterpret_cast<half *>(reinterpret_cast<uint8_t *>(workspace) + DYNAMIC_WORKSPACE_SIZE);
                        auto stream = hipStreamLegacy;
                        {
                            auto itemsPerLine = info.headDim * sizeof(half) / sizeof(float4);
                            auto threads = info.batch * info.nHead * info.seqLen * itemsPerLine;
                            auto blocks = (threads + 1023) / 1024;
                            concatCache<<<blocks, 1024, 0, stream>>>(
                                kCache, k,
                                info.seqLen * itemsPerLine,
                                info.cacheLen * itemsPerLine,
                                past * itemsPerLine,
                                threads);
                            concatCache<<<blocks, 1024, 0, stream>>>(
                                vCache, v,
                                info.seqLen * itemsPerLine,
                                info.cacheLen * itemsPerLine,
                                past * itemsPerLine,
                                threads);
                        }
                        MatrixDescriptor
                            q_(MatrixLayout{
                                .dataType = dataTypeConvert(info.dataType),
                                .rows = static_cast<uint64_t>(info.seqLen),
                                .cols = static_cast<uint64_t>(info.headDim),
                                .majorStride = static_cast<int64_t>(info.headDim),
                                .order = ROW_MAJOR,
                                .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                                .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                            }),
                            k_(MatrixLayout{
                                .dataType = dataTypeConvert(info.dataType),
                                .rows = static_cast<uint64_t>(info.headDim),
                                .cols = static_cast<uint64_t>(attLen),
                                .majorStride = static_cast<int64_t>(info.headDim),
                                .order = COL_MAJOR,
                                .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                                .batchStride = static_cast<int64_t>(info.cacheLen * info.headDim),
                            }),
                            v_(MatrixLayout{
                                .dataType = dataTypeConvert(info.dataType),
                                .rows = static_cast<uint64_t>(attLen),
                                .cols = static_cast<uint64_t>(info.headDim),
                                .majorStride = static_cast<int64_t>(info.headDim),
                                .order = ROW_MAJOR,
                                .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                                .batchStride = static_cast<int64_t>(info.cacheLen * info.headDim),
                            }),
                            att_(MatrixLayout{
                                .dataType = dataTypeConvert(info.dataType),
                                .rows = static_cast<uint64_t>(info.seqLen),
                                .cols = static_cast<uint64_t>(attLen),
                                .majorStride = static_cast<int64_t>(info.cacheLen),
                                .order = ROW_MAJOR,
                                .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                                .batchStride = static_cast<int64_t>(info.cacheLen * info.seqLen),
                            });
                        {
                            auto [algo, workspaceSize] = tune(
                                handle, d->mul,
                                q_, k_, att_,
                                DYNAMIC_WORKSPACE_SIZE);
                            half alpha = rsqrtf(info.headDim), beta = 0;
                            hipblasLtMatmul(
                                handle, d->mul.get(),
                                &alpha,
                                q, q_.get(),
                                kCache, k_.get(),
                                &beta,
                                att, att_.get(),
                                att, att_.get(),
                                &algo,
                                workspace, workspaceSize,
                                stream);
                        }
                        softmax<<<dim3(info.batch * info.nHead, info.seqLen),
                                  std::min(1024u, attLen),
                                  attLen * sizeof(float),
                                  stream>>>(
                            att, AttentionCausualMask(), attLen, info.cacheLen);
                        {
                            auto [algo, workspaceSize] = tune(
                                handle, d->mul,
                                att_, v_, q_,
                                DYNAMIC_WORKSPACE_SIZE);
                            half alpha = 1, beta = 0;
                            hipblasLtMatmul(
                                handle, d->mul.get(),
                                &alpha,
                                att, att_.get(),
                                vCache, v_.get(),
                                &beta,
                                o, q_.get(),
                                o, q_.get(),
                                &algo,
                                workspace, workspaceSize,
                                stream);
                        }
                    };

                return {std::move(routine), workspaceSize};
            }
            TODO("");
        }

        TODO("");
    }

}// namespace refactor::kernel
