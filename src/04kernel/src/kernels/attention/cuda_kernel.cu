#include "hip/hip_runtime.h"
﻿#include "../../utilities/cuda/cublaslt_utils.cuh"
#include "cuda_kernel.hh"
#include "hardware/functions.h"
#include "kernel/cuda/reduce.cuh"

namespace refactor::kernel {
    using K = AttentionCuda;
    using namespace hipcublas;

    static __forceinline__ __device__ bool mask(int tokid, int posid) {
        return true;
    }

    // gridDim.x = batch * nHead
    // gridDim.y = seqLen
    template<class T, class Mask>
    static __global__ void softmax(
        T *__restrict__ attention,
        Mask mask,
        uint32_t seqLen,
        uint32_t bufLen) {
        // int offset = (blockIdx.x * len_q + blockIdx.y) * len_buf;
        // SharedMemory<float> shared;
        // float *smem = shared.getPointer();

        // for (int i = threadIdx.x; i < len_buf; i += blockDim.x) {
        //     T pb = (position_bias == nullptr) ? T(0.) : position_bias[offset + i];
        //     smem[i] = mask[blockIdx.y * len_buf + i] > 0 ? x[offset + i] * scale + pb : -Inf<T>();
        // }
        // float local_max = -1e20;
        // for (int i = threadIdx.x; i < len_buf; i += blockDim.x) {
        //     local_max = fmaxf(local_max, smem[i]);
        // }
        // local_max = functions::blockReduceMax<float>(local_max);

        // float local_sum = 1e-20;
        // for (int i = threadIdx.x; i < len_buf; i += blockDim.x) {
        //     float v = expf(float(smem[i]) - local_max);
        //     smem[i] = v;
        //     local_sum += v;
        // }
        // local_sum = functions::blockReduceSum<float>(local_sum);
        // for (int i = threadIdx.x; i < len_buf; i += blockDim.x) {
        //     x[offset + i] = float(smem[i]) / local_sum;
        // }
    }

    RoutineWorkspace K::lower(Resources &res) const {
        auto handle = res.fetchOrStore<CublasLtContext>()->handle;

        constexpr auto ROW_MAJOR = HIPBLASLT_ORDER_ROW;
        constexpr auto COL_MAJOR = HIPBLASLT_ORDER_COL;

        if (!info.cacheLen) {
            if (info.nHead == info.nKVHead) {
                // RAII for closure
                struct Descriptors {
                    MatMulDescriptor mul;
                    MatrixDescriptor q, k, v, att;
                    hipblasLtMatmulAlgo_t algoQK, algoAV;
                    size_t attSize, workspaceSizeQK, workspaceSizeAV;

                    Descriptors(CublasLtContext const &context,
                                AttentionInfo info)
                        : mul(computeTypeConvert(info.dataType),
                              dataTypeConvert(info.dataType)),
                          q(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.headDim),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          k(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.headDim),
                              .cols = static_cast<uint64_t>(info.seqLen),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = COL_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          v(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.headDim),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          att(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.seqLen),
                              .majorStride = static_cast<int64_t>(info.seqLen),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.seqLen),
                          }),
                          attSize(info.batch * info.nHead * info.seqLen * info.seqLen * info.dataType.size()) {
                        auto [algoQK_, workspaceSizeQK_] = tune(context.handle, mul, q, k, att);
                        auto [algoAV_, workspaceSizeAV_] = tune(context.handle, mul, att, v, q);
                        algoQK = algoQK_;
                        algoAV = algoAV_;
                        workspaceSizeQK = workspaceSizeQK_;
                        workspaceSizeAV = workspaceSizeAV_;
                    }
                };

                auto const &context = *res.fetchOrStore<CublasLtContext>();
                auto d = std::make_shared<Descriptors>(context, info);
                auto workspaceSize = d->attSize;
                workspaceSize = hardware::alignBytes(workspaceSize, 256);
                workspaceSize += d->workspaceSizeQK;
                workspaceSize += d->workspaceSizeAV;
                workspaceSize = hardware::alignBytes(workspaceSize, 256);

                auto routine = [d = std::move(d), info = this->info]//
                    (Resources & res, void *workspace, void const *const *inputs, void *const *outputs) {
                        auto handle = res.fetchOrStore<CublasLtContext>()->handle;
                        auto q = inputs[0];
                        auto k = inputs[1];
                        auto v = inputs[2];
                        auto o = outputs[0];
                        auto att = workspace;
                        auto workspaceQK = reinterpret_cast<uint8_t *>(workspace) + hardware::alignBytes(d->attSize, 256);
                        auto workspaceAV = workspaceQK + hardware::alignBytes(d->workspaceSizeQK, 256);

                        float alpha = 1, beta = 0;
                        hipblasLtMatmul(
                            handle, d->mul.get(),
                            &alpha,
                            q, d->q.get(),
                            k, d->k.get(),
                            &beta,
                            att, d->att.get(),
                            att, d->att.get(),
                            &d->algoQK,
                            workspaceQK, d->workspaceSizeQK,
                            hipStreamLegacy);

                        softmax<<<dim3(info.batch * info.nHead, info.seqLen), info.seqLen>>>(
                            att, mask, info.seqLen, info.seqLen);

                        hipblasLtMatmul(
                            handle, d->mul.get(),
                            &alpha,
                            att, d->att.get(),
                            v, d->v.get(),
                            &beta,
                            o, d->q.get(),
                            o, d->q.get(),
                            &d->algoAV,
                            workspaceAV, d->workspaceSizeAV,
                            hipStreamLegacy);
                    };
                return {std::move(routine), workspaceSize};
            }
        }
        TODO("");
    }

}// namespace refactor::kernel
