﻿#include "../../utilities/cuda/cublaslt_utils.cuh"
#include "cuda_kernel.hh"
#include "hardware/functions.h"
#include "kernel/cuda/reduce.cuh"

namespace refactor::kernel {
    using K = AttentionCuda;
    using namespace hipcublas;

    RoutineWorkspace K::lower(Resources &res) const {
        auto handle = res.fetchOrStore<CublasLtContext>()->handle;

        constexpr auto ROW_MAJOR = HIPBLASLT_ORDER_ROW;
        constexpr auto COL_MAJOR = HIPBLASLT_ORDER_COL;

        if (!info.cacheLen) {
            if (info.nHead == info.nKVHead) {
                // RAII for closure
                struct Descriptors {
                    MatMulDescriptor mul;
                    MatrixDescriptor q, k, v, att;
                    hipblasLtMatmulAlgo_t algoQK, algoAV;
                    size_t attSize, workspaceSizeQK, workspaceSizeAV;

                    Descriptors(CublasLtContext const &context,
                                hipblasComputeType_t compute,
                                AttentionInfo info)
                        : mul(compute, HIP_R_32F),
                          q(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.headDim),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          k(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.headDim),
                              .cols = static_cast<uint64_t>(info.seqLen),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = COL_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          v(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.headDim),
                              .majorStride = static_cast<int64_t>(info.headDim),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.headDim),
                          }),
                          att(MatrixLayout{
                              .dataType = dataTypeConvert(info.dataType),
                              .rows = static_cast<uint64_t>(info.seqLen),
                              .cols = static_cast<uint64_t>(info.seqLen),
                              .majorStride = static_cast<int64_t>(info.seqLen),
                              .order = ROW_MAJOR,
                              .batchCount = static_cast<int32_t>(info.batch * info.nHead),
                              .batchStride = static_cast<int64_t>(info.seqLen * info.seqLen),
                          }),
                          attSize(info.batch * info.nHead * info.seqLen * info.seqLen * info.dataType.size()) {
                        auto [algoQK_, workspaceSizeQK_] = tune(context.handle, mul, q, k, att);
                        auto [algoAV_, workspaceSizeAV_] = tune(context.handle, mul, att, v, q);
                        algoQK = algoQK_;
                        algoAV = algoAV_;
                        workspaceSizeQK = workspaceSizeQK_;
                        workspaceSizeAV = workspaceSizeAV_;
                    }
                };

                auto const &context = *res.fetchOrStore<CublasLtContext>();
                auto d = std::make_shared<Descriptors>(context, HIPBLAS_COMPUTE_32F, info);
                auto workspaceSize = d->attSize;
                workspaceSize = hardware::alignBytes(workspaceSize, 256);
                workspaceSize += d->workspaceSizeQK;
                workspaceSize = hardware::alignBytes(workspaceSize, 256);
                workspaceSize += d->workspaceSizeAV;
                workspaceSize = hardware::alignBytes(workspaceSize, 256);

                auto routine = [d = std::move(d), info = this->info]//
                    (Resources & res, void *workspace, void const *const *inputs, void *const *outputs) {
                        auto handle = res.fetchOrStore<CublasLtContext>()->handle;
                        auto q = inputs[0];
                        auto k = inputs[1];
                        auto v = inputs[2];
                        auto o = outputs[0];
                        auto att = workspace;
                        auto workspaceQK = reinterpret_cast<uint8_t *>(workspace) + hardware::alignBytes(d->attSize, 256);
                        auto workspaceAV = workspaceQK + hardware::alignBytes(d->workspaceSizeQK, 256);

                        float alpha = 1, beta = 0;
                        hipblasLtMatmul(
                            handle, d->mul.get(),
                            &alpha,
                            q, d->q.get(),
                            k, d->k.get(),
                            &beta,
                            att, d->att.get(),
                            att, d->att.get(),
                            &d->algoQK,
                            workspaceQK, d->workspaceSizeQK,
                            hipStreamLegacy);

                        // TODO inline mask && softmax

                        hipblasLtMatmul(
                            handle, d->mul.get(),
                            &alpha,
                            att, d->att.get(),
                            v, d->v.get(),
                            &beta,
                            o, d->q.get(),
                            o, d->q.get(),
                            &d->algoAV,
                            workspaceAV, d->workspaceSizeAV,
                            hipStreamLegacy);
                    };
                return {std::move(routine), workspaceSize};
            }
        }
        TODO("");
    }

}// namespace refactor::kernel
