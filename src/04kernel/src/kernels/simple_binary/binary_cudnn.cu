#include "../../utilities/cuda/cudnn_context.hh"
#include "../../utilities/cuda/cudnn_functions.h"
#include "binary_cudnn.hh"
#include "kernel/kernel.h"
#include "kernel/tensor.h"

namespace refactor::kernel {
    using namespace cudnn;
    using namespace runtime;
    Routine BinaryCudnn::lower() const noexcept {
        struct Descriptors {
            hipdnnOpTensorDescriptor_t opDesc;
            hipdnnTensorDescriptor_t aDesc, bDesc, cDesc;
            float aAlpha = 1.f;
            float bAlpha = 1.f;
            float beta = 0.f;
            Descriptors() {
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&aDesc));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&bDesc));
                CUDNN_ASSERT(hipdnnCreateTensorDescriptor(&cDesc));
                CUDNN_ASSERT(hipdnnCreateOpTensorDescriptor(&opDesc));
            }
            ~Descriptors() {
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(aDesc));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(bDesc));
                CUDNN_ASSERT(hipdnnDestroyTensorDescriptor(cDesc));
                CUDNN_ASSERT(hipdnnDestroyOpTensorDescriptor(opDesc));
            }
        };
        auto d = std::make_shared<Descriptors>();
        hipdnnOpTensorOp_t cudnnOP;
        if (opType == SimpleBinaryType::Add) {
            cudnnOP = HIPDNN_OP_TENSOR_ADD;
        } else if (opType == SimpleBinaryType::Sub) {
            cudnnOP = HIPDNN_OP_TENSOR_ADD;
            d->bAlpha = -1.f;
        } else if (opType == SimpleBinaryType::Mul) {
            cudnnOP = HIPDNN_OP_TENSOR_MUL;
        }

        setCudnnTensor(d->aDesc, dataType, aDims);
        setCudnnTensor(d->bDesc, dataType, bDims);
        setCudnnTensor(d->cDesc, dataType, cDims);
        CUDNN_ASSERT(hipdnnSetOpTensorDescriptor(
            d->opDesc, cudnnOP, cudnnDataTypeConvert(dataType), HIPDNN_NOT_PROPAGATE_NAN));

        bool swap = aDims != cDims;

        return [d_ = std::move(d), swap](Resources &res, void const **inputs, void **outputs) {
            auto handle = res.fetchOrStore<CudnnContext>()->handle;
            auto const &d = *d_;
            // name inputs and outputs
            auto a = inputs[0],
                 b = inputs[1];
            auto c = outputs[0];
            if (swap) {
                CUDNN_ASSERT(hipdnnOpTensor(handle, d.opDesc, &(d.aAlpha),
                                           d.bDesc, b, &(d.bAlpha), d.aDesc, a,
                                           &(d.beta), d.cDesc, c));
            } else {
                CUDNN_ASSERT(hipdnnOpTensor(handle, d.opDesc, &(d.aAlpha),
                                           d.aDesc, a, &(d.bAlpha), d.bDesc, b,
                                           &(d.beta), d.cDesc, c));
            }
        };
    }
}// namespace refactor::kernel
