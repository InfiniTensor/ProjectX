#include "hip/hip_runtime.h"
﻿#include "no_broadcast_cuda.hh"

namespace refactor::kernel {
    using K = Binary11Cuda;
    using Op = SimpleBinaryType;
    using DT = DataType;

#define KERNEL(NAME, OP)                                                        \
    template<typename T>                                                        \
    __global__ void _kernel_##NAME(T const *a, T const *b, T *c, size_t size) { \
        size_t tid = threadIdx.x + blockIdx.x * blockDim.x;                     \
        if (tid < size) {                                                       \
            c[tid] = a[tid] OP b[tid];                                          \
        }                                                                       \
    }

    KERNEL(Add, +)
    KERNEL(Sub, -)
    KERNEL(Mul, *)
    KERNEL(Div, /)

#define CASE_DT(NAME, T)                                                                     \
    case DT::T:                                                                              \
        return [n = this->size](runtime::Resources &, void const **inputs, void **outputs) { \
            using T_ = primitive_t<DT::T>::type;                                             \
            auto a = static_cast<T_ const *>(inputs[0]);                                     \
            auto b = static_cast<T_ const *>(inputs[1]);                                     \
            auto c = static_cast<T_ *>(outputs[0]);                                          \
            size_t blocksize = 1024;                                                         \
            size_t gridsize = (n + blocksize - 1) / blocksize;                               \
            _kernel_##NAME<<<gridsize, blocksize>>>(a, b, c, n);                             \
        }

#define CASE_OP(NAME)                \
    case Op::NAME:                   \
        switch (dataType.internal) { \
            CASE_DT(NAME, F32);      \
            CASE_DT(NAME, U8);       \
            CASE_DT(NAME, I8);       \
            CASE_DT(NAME, U16);      \
            CASE_DT(NAME, I16);      \
            CASE_DT(NAME, I32);      \
            CASE_DT(NAME, I64);      \
            CASE_DT(NAME, F64);      \
            CASE_DT(NAME, U32);      \
            CASE_DT(NAME, U64);      \
            default:                 \
                UNREACHABLE();       \
        }

    auto K::lower() const noexcept -> Routine {
        switch (opType) {
            CASE_OP(Add)
            CASE_OP(Sub)
            CASE_OP(Mul)
            CASE_OP(Div)
            default:
                UNREACHABLE();
        }
    }
}// namespace refactor::kernel
