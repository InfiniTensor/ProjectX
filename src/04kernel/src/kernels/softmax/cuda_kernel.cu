#include "hip/hip_runtime.h"
#include "cuda_kernel.hh"
#include <hipcub/hipcub.hpp>

namespace refactor::kernel {
    using namespace runtime;

    template<class T>
    struct MD {// update the global max and sum, store the output at
               // max and sum
        T max; // store max
        T sum; // store sum
    };
    template<class T>
    __device__ __forceinline__ MD<T> reduce_md_op(MD<T> a, MD<T> b) {
        if (a.max > b.max) {
            return {a.max, a.sum + b.sum * __expf(b.max - a.max)};
        } else {
            return {b.max, b.sum + a.sum * __expf(a.max - b.max)};
        }
    }

    template<int BLOCK_DIM, class T>
    __launch_bounds__(BLOCK_DIM) __global__ void _blockSoftmaxKernel(
        T const *__restrict input, T *__restrict output, int size, int dimsize, int stride) {
        // if set axis = 1, inputShape=[I,J,K,S]
        // tid = i(JKS) + j(KS) + k(S) + s

        // blockDim.x = size/dimsize = IKS
        // blockIdx.x = i(KS) + k(S) + s,blockIdx.x%stride = k(S) + s

        // now, tid = i(JKS) + k(S) + s;
        int tid = blockIdx.x % stride + (blockIdx.x - blockIdx.x % stride) * dimsize;

        MD<T> mdPartial{-__FLT_MAX__, 0};
        for (int i = threadIdx.x; i < dimsize; i += BLOCK_DIM) {
            mdPartial = reduce_md_op(mdPartial, {input[tid + i * stride], 1});// reduce the data to one block
        }
        using BlockReduce = hipcub::BlockReduce<MD<T>, BLOCK_DIM>;
        __shared__ typename BlockReduce::TempStorage tempStorage;
        __shared__ MD<T> mdTotal;
        MD<T> mdBlock = BlockReduce(tempStorage).Reduce(mdPartial, reduce_md_op<T>);
        if (threadIdx.x == 0) {
            mdTotal = mdBlock;// must set threadIdx.x = 0 write the output to memory
        }
        __syncthreads();

        for (int i = threadIdx.x; i < dimsize; i += BLOCK_DIM) {
            auto j = tid + i * stride;
            output[j] = __expf(input[j] - mdTotal.max) * __fdividef(1, mdTotal.sum);
        }
    }

    template<class T> struct SumOp {
        __device__ __forceinline__ T operator()(const T &a, const T &b) const {
            return a + b;
        }
    };
    template<class T> struct MaxOp {
        __device__ __forceinline__ T operator()(const T &a, const T &b) const {
            return max(a, b);
        }
    };
    template<class ReductionOp, class T>
    __device__ __forceinline__ T WarpAllReduce(T val, int threadGroupWidth, ReductionOp op) {
        for (int mask = threadGroupWidth / 2; mask > 0; mask /= 2) {
            val = op(val, __shfl_xor_sync(0xffffffff, val, mask));
        }
        return val;
    }

    template<class T>
    __global__ void _warpSoftmaxKernel(
        T const *__restrict input,
        T *__restrict output,
        int size, int dimsize, int stride) {

        int otherIdx = blockIdx.x * blockDim.y + threadIdx.y;
        int tid = otherIdx % stride + (otherIdx - otherIdx % stride) * dimsize;

        extern __shared__ char shared[];
        if (otherIdx < size / dimsize) {
            auto maxTotal = reinterpret_cast<T *>(shared),
                 sumTotal = maxTotal + blockDim.y;

            T maxData = -__FLT_MAX__;
            for (int i = threadIdx.x; i < dimsize; i += blockDim.x) {
                maxData = max(maxData, input[tid + i * stride]);
            }
            maxData = WarpAllReduce(maxData, blockDim.x, MaxOp<T>{});
            if (threadIdx.x == 0) {
                maxTotal[threadIdx.y] = maxData;
            }

            //--------------------------------------------
            T sumData = 0;
            for (int i = threadIdx.x; i < dimsize; i += blockDim.x) {
                sumData += __expf(input[tid + i * stride] - maxTotal[threadIdx.y]);
            }
            sumData = WarpAllReduce(sumData, blockDim.x, SumOp<T>{});
            if (threadIdx.x == 0) {
                sumTotal[threadIdx.y] = sumData;
            }

            //--------------------------------------------
            for (int i = threadIdx.x; i < dimsize; i += blockDim.x) {
                auto j = tid + i * stride;
                output[j] = __expf(input[j] - maxTotal[threadIdx.y]) * __fdividef(1, sumTotal[threadIdx.y]);
            }
        }
    }

    template<class T>
    Routine lowerTypedCuda(SoftmaxInfo info) {
        using namespace runtime;

        return [info](Resources &, void *workspace, void const *const *inputs, void *const *outputs) {
            auto x = reinterpret_cast<T const *>(inputs[0]);
            auto y = reinterpret_cast<T *>(outputs[0]);
            int numBlocks = info.pre * info.post,
                dimsize = info.mid,
                size = numBlocks * dimsize,
                stride = info.post;
            if (dimsize > 1024) {
                _blockSoftmaxKernel<1024><<<numBlocks, 1024>>>(x, y, size, dimsize, stride);
            } else {
                // clang-format off
                int blockDimX = dimsize > 31 ? 32
                              : dimsize > 15 ? 16
                              : dimsize >  7 ?  8
                                             :  4,
                    blockDimY = 1024 / blockDimX;
                // clang-format on
                _warpSoftmaxKernel<<<(numBlocks + blockDimY - 1) / blockDimY,
                                     dim3(blockDimX, blockDimY),
                                     blockDimY * 2 * sizeof(T)>>>(x, y, size, dimsize, stride);
            }
        };
    }

    auto SoftmaxCuda::lower(Resources &res) const noexcept -> RoutineWorkspace {
        switch (info.type.internal) {
            case DataType::F32:
                return lowerTypedCuda<float>(info);
            case DataType::F64:
                return lowerTypedCuda<double>(info);
            // case DataType::FP16:
            //     return lowerTypedCuda<half>(info);
            default:
                UNREACHABLE();
        }
    }

}// namespace refactor::kernel
