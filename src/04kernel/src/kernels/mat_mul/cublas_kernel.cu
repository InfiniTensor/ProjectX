﻿#include "../../utilities/cuda/cublas_context.hh"
#include "../expand/cuda_kernel.hh"
#include "cublas_kernel.hh"

namespace refactor::kernel {
    using namespace runtime;
    using namespace hipcublas;

    template<class T>
    static auto lowerTyped(hipDataType hipDataType,
                           MatMulInfo info,
                           Resources &res) noexcept -> RoutineWorkspace {
        // clang-format off
        auto alpha   = static_cast<T>(info.alpha),
             beta    = static_cast<T>(info.biasExpand ? info.beta : 0.0f);
        auto tA      = info.transA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
             tB      = info.transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        auto m       = info.m,
             n       = info.n,
             k       = info.k;
        auto strideY = m * n,
             strideA = m * k,
             strideB = k * n;
        auto lda     = info.transA ? m : k,
             ldb     = info.transB ? k : n;
        auto biasEx  = info.biasExpand
                     ? std::make_optional(ExpandCuda(*info.biasExpand).lower(res).routine)
                     : std::nullopt;
        // clang-format on
        if (info.broadcaster.needBroadcast()) {
            return [broadcaster = info.broadcaster,
                    hipDataType,
                    alpha, beta, tA, tB,
                    m, n, k,
                    strideY, strideA, strideB,
                    lda, ldb,
                    biasEx]//
                (Resources & res, void *workspace, void const *const *inputs, void *const *outputs) {
                    if (biasEx) { (*biasEx)(res, workspace, inputs + 2, outputs); }

                    auto handle = res.fetchOrStore<CublasContext>()->handle;
                    auto a = reinterpret_cast<T const *>(inputs[0]);
                    auto b = reinterpret_cast<T const *>(inputs[1]);
                    auto y = reinterpret_cast<T *>(outputs[0]);
                    uint32_t offset[2];
                    for (auto i : range0_(broadcaster.outputsCount)) {
                        broadcaster.locate(i, offset);
                        CUBLAS_ASSERT(hipblasGemmEx(
                            handle,
                            tB, tA, n, m, k,
                            &alpha,
                            b + strideB * offset[1], hipDataType, ldb,
                            a + strideA * offset[0], hipDataType, lda,
                            &beta, y + strideY * i, hipDataType, n,
                            hipDataType, HIPBLAS_GEMM_DEFAULT));
                    }
                };

        } else {
            return [batch = info.broadcaster.outputsCount,
                    hipDataType,
                    alpha, beta, tA, tB,
                    m, n, k,
                    strideY, strideA, strideB,
                    lda, ldb,
                    biasEx]//
                (Resources & res, void *workspace, void const *const *inputs, void *const *outputs) {
                    // Call expand kernel to broadcast bias if bias is used
                    if (biasEx) { (*biasEx)(res, workspace, inputs + 2, outputs); }

                    auto handle = res.fetchOrStore<CublasContext>()->handle;
                    auto a = reinterpret_cast<T const *>(inputs[0]);
                    auto b = reinterpret_cast<T const *>(inputs[1]);
                    auto y = reinterpret_cast<T *>(outputs[0]);
                    CUBLAS_ASSERT(hipblasGemmStridedBatchedEx(
                        handle,
                        tB, tA, n, m, k,
                        &alpha,
                        b, hipDataType, ldb, strideB,
                        a, hipDataType, lda, strideA,
                        &beta, y, hipDataType, n,
                        strideY, batch,
                        hipDataType, HIPBLAS_GEMM_DEFAULT));
                };
        }
    }

    auto MatMulCublas::lower(Resources &res) const noexcept -> RoutineWorkspace {
        res.fetchOrStore<CublasContext>();
        switch (info.dataType) {
            case DataType::F32:
                return lowerTyped<float>(HIP_R_32F, info, res);
            case DataType::F64:
                return lowerTyped<double>(HIP_R_64F, info, res);
            case DataType::FP16:
                return lowerTyped<half>(HIP_R_16F, info, res);
            default:
                UNREACHABLE();
        }
    }

}// namespace refactor::kernel
