﻿#include "../../utilities/cuda/cublas_context.hh"
#include "cublas_kernel.hh"
#include <hipblas.h>
#include <thrust/execution_policy.h>
#include <thrust/tabulate.h>

namespace refactor::kernel {
    using namespace runtime;
    using namespace hipcublas;

    template<class T>
    struct MatMulBroadcastBiasFunctor {
        T const *src;
        size_t const n, strideC0, strideC1;

        __device__ T operator()(size_t i) const noexcept {
            return src[i / n * strideC0 + i % n * strideC1];
        }
    };

    template<class T>
    struct MatMulCopyBiasFunctor {
        T const *src;
        size_t blockSize;

        __device__ T operator()(size_t i) const noexcept {
            return src[i % blockSize];
        }
    };

    template<class T>
    Routine lowerTyped(hipDataType hipDataType, MatMulInfo info, size_t strideC0, size_t strideC1) noexcept {
        return [hipDataType,
                alpha = static_cast<T>(info.alpha),
                beta = static_cast<T>(info.biasType != BiasType::NoBias ? info.beta : 0.0f),
                tA = info.transA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                tB = info.transB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                m = info.m, n = info.n, k = info.k,
                strideY = info.m * info.n,
                strideA = info.m * info.k,
                strideB = info.k * info.n,
                strideC0, strideC1,
                lda = info.transA ? info.m : info.k,
                ldb = info.transB ? info.k : info.n,
                broadcaster = info.broadcaster](Resources &res, void const **inputs, void **outputs) {
            auto a = reinterpret_cast<T const *>(inputs[0]);
            auto b = reinterpret_cast<T const *>(inputs[1]);
            auto y = reinterpret_cast<T *>(outputs[0]);

            if (beta != (T) 0) {
                // Expand bias to 2D and store in final output Y
                {
                    auto c = reinterpret_cast<T const *>(inputs[2]);
                    thrust::tabulate(
                        thrust::device,
                        y,
                        y + strideY,
                        MatMulBroadcastBiasFunctor<T>{c, n, strideC0, strideC1});
                }
                // Copy 2D bias to each batch
                if (broadcaster.outputsCount > 1) {
                    thrust::tabulate(
                        thrust::device,
                        y + strideY,
                        y + strideY * broadcaster.outputsCount,
                        MatMulCopyBiasFunctor<T>{y, strideY});
                }
            }

            auto handle = res.fetchOrStore<CublasContext>()->handle;
            uint32_t offset[2];
            for (auto i : range0_(broadcaster.outputsCount)) {
                broadcaster.locate(i, offset);
                auto stat = hipblasGemmEx(
                    handle, tB, tA, n, m, k, &alpha, b + strideB * offset[1],
                    hipDataType, ldb, a + strideA * offset[0], hipDataType, lda, &beta, y + strideY * i,
                    hipDataType, n, hipDataType, HIPBLAS_GEMM_DEFAULT);
            }
        };
    }

    Routine MatMulCublas::lower(Resources &res) const noexcept {
        size_t strideC0 = 0, strideC1 = 0;
        switch (info.biasType) {
            case BiasType::NoBias:
            case BiasType::Scalar:
                break;
            case BiasType::RowVector:
                strideC1 = 1;
                break;
            case BiasType::ColVector:
                strideC0 = 1;
                break;
            case BiasType::Matrix:
                strideC1 = 1;
                strideC0 = info.n;
                break;
            default:
                UNREACHABLE();
        }

        res.fetchOrStore<CublasContext>();
        switch (info.dataType) {
            case DataType::F32:
                return lowerTyped<float>(HIP_R_32F, info, strideC0, strideC1);
            case DataType::F64:
                return lowerTyped<double>(HIP_R_64F, info, strideC0, strideC1);
            case DataType::FP16:
                return lowerTyped<half>(HIP_R_16F, info, strideC0, strideC1);
            default:
                UNREACHABLE();
        }
    }

}// namespace refactor::kernel
