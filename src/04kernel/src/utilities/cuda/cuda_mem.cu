﻿#include "cuda_mem.h"
#include <hip/hip_runtime.h>

namespace refactor::kernel::cuda {

    void *malloc(size_t bytes) {
        void *ans;
        hipMalloc(&ans, bytes);
        return ans;
    }
    void free(void *ptr) {
        hipFree(ptr);
    }
    void *memcpy_h2d(void *dst, void const *src, size_t bytes) noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyHostToDevice);
        return dst;
    }
    void *memcpy_d2h(void *dst, void const *src, size_t bytes) noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyDeviceToHost);
        return dst;
    }
    void *memcpy_d2d(void *dst, void const *src, size_t bytes) noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyDeviceToDevice);
        return dst;
    }

    mem_manager::MemFunctions const &memFunc() {
        static mem_manager::MemFunctions F{
            malloc,
            free,
            memcpy_h2d,
            memcpy_d2h,
            memcpy_d2d,
        };
        return F;
    }


}// namespace refactor::kernel::cuda
