﻿#include "cuda_mem.cuh"
#include <hip/hip_runtime.h>

namespace refactor::kernel::cuda {

    Arc<hardware::MemManager> BasicCudaMemManager::instance() {
        static auto I = std::make_shared<BasicCudaMemManager>();
        return I;
    }
    void *BasicCudaMemManager::malloc(size_t bytes) noexcept {
        void *ans;
        hipMalloc(&ans, bytes);
        return ans;
    }
    void BasicCudaMemManager::free(void *ptr) noexcept {
        hipFree(ptr);
    }
    void *BasicCudaMemManager::copyHD(void *dst, void const *src, size_t bytes) const noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyHostToDevice);
        return dst;
    }
    void *BasicCudaMemManager::copyDH(void *dst, void const *src, size_t bytes) const noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyDeviceToHost);
        return dst;
    }
    void *BasicCudaMemManager::copyDD(void *dst, void const *src, size_t bytes) const noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyDeviceToDevice);
        return dst;
    }

}// namespace refactor::kernel::cuda
