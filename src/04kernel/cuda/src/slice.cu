#include "hip/hip_runtime.h"
﻿#include "kernel/cuda/slice.cuh"
#include "macro.cuh"
#include <cstdint>

namespace refactor::kernel::cuda {

    __global__ static void sliceKernel(
        unsigned long long n,
        uint8_t const *src, DimInfo const *dims, uint8_t *output,
        unsigned int rank,
        unsigned int blockSize) {
        extern __shared__ DimInfo dimInfo[];
        for (auto i = threadIdx.x; i < rank; i += blockDim.x) {
            dimInfo[i] = dims[i];
        }
        __syncthreads();
        for (auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                  step = blockDim.x * gridDim.x;
             tid < n;
             tid += step) {
            long rem = tid;
            auto src_ = src;
            auto dst_ = output + rem * blockSize;
            for (auto i = 0; i < rank; ++i) {
                auto const &dim = dimInfo[i];
                src_ += rem / dim.countStride * dim.sizeStride + dim.sizeStart;
                rem %= dim.countStride;
            }
            optimizedMemcpy(dst_, src_, blockSize);
        }
    }

    void launchSlice(
        KernelLaunchParameters const &params,
        void const *src, DimInfo const *dims, void *output,
        unsigned int rank,
        unsigned int blockSize) {
        sliceKernel<<<
            params.gridSize,
            params.blockSize,
            rank * sizeof(DimInfo),
            reinterpret_cast<hipStream_t>(params.stream)>>>(
            params.n,
            reinterpret_cast<uint8_t const *>(src),
            dims,
            reinterpret_cast<uint8_t *>(output),
            rank,
            blockSize);
    }

}// namespace refactor::kernel::cuda
