#include "hip/hip_runtime.h"
﻿#include "kernel/cuda/transpose.cuh"
#include "macro.cuh"
#include <cstdint>

namespace refactor::kernel::cuda {

    __global__ static void transposeKernel(
        unsigned long long n,
        uint8_t const *__restrict__ data,
        transpose::DimStride const *__restrict__ strides,
        uint8_t *__restrict__ output,
        unsigned int rank,
        unsigned int eleSize) {
        for (auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                  step = blockDim.x * gridDim.x;
             tid < n;
             tid += step) {
            auto j = 0u, rem = tid;
            for (auto k = 0u; k < rank; ++k) {
                auto o_ = __ldg(&(strides[k].o));
                auto i_ = __ldg(&(strides[k].i));
                j += rem / o_ * i_;
                rem %= o_;
            }

            optimizedMemcpy(output + tid * eleSize, data + j * eleSize, eleSize);
        }
    }

    void launchTranspose(
        KernelLaunchParameters const &params,
        void const *data, transpose::DimStride const *strides, void *output,
        unsigned int rank,
        unsigned int eleSize) {
        transposeKernel<<<
            params.gridSize,
            params.blockSize,
            0,
            reinterpret_cast<hipStream_t>(params.stream)>>>(
            params.n,
            reinterpret_cast<uint8_t const *>(data),
            strides,
            reinterpret_cast<uint8_t *>(output),
            rank,
            eleSize);
    }

}// namespace refactor::kernel::cuda
