#include "hip/hip_runtime.h"
﻿#include "kernel/cuda/where.cuh"
#include "macro.cuh"
#include <cstdint>

namespace refactor::kernel::cuda {

    __global__ static void whereKernel(
        unsigned long long n,
        unsigned int const *strides,
        bool const *c,
        uint8_t const *x,
        uint8_t const *y,
        uint8_t *output,
        unsigned int rank,
        unsigned int eleSize) {
        extern __shared__ unsigned int shared[];
        for (auto i = threadIdx.x; i < rank * 4; i += blockDim.x) {
            shared[i] = strides[i];
        }
        __syncthreads();
        for (auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                  step = blockDim.x * gridDim.x;
             tid < n;
             tid += step) {
            auto ic = 0u, ix = 0u, iy = 0u, rem = tid;
            for (auto j = 0u; j < rank; ++j) {
                auto dim = shared + 4 * j;
                auto quot = rem / dim[3];
                rem %= dim[3];
                ic += quot * dim[0];
                ix += quot * dim[1];
                iy += quot * dim[2];
            }

            optimizedMemcpy(output + tid * eleSize,
                            c[ic]
                                ? x + ix * eleSize
                                : y + iy * eleSize,
                            eleSize);
        }
    }

    void launchWhere(
        KernelLaunchParameters const &params,
        unsigned int const *strides,
        void const *c,
        void const *x,
        void const *y,
        void *output,
        unsigned int rank,
        unsigned int eleSize) {
        whereKernel<<<
            params.gridSize,
            params.blockSize,
            rank * sizeof(unsigned int) * 4,
            reinterpret_cast<hipStream_t>(params.stream)>>>(
            params.n,
            strides,
            reinterpret_cast<bool const *>(c),
            reinterpret_cast<uint8_t const *>(x),
            reinterpret_cast<uint8_t const *>(y),
            reinterpret_cast<uint8_t *>(output),
            rank,
            eleSize);
    }

}// namespace refactor::kernel::cuda
