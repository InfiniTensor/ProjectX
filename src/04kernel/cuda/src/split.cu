#include "hip/hip_runtime.h"
﻿#include "kernel/cuda/split.cuh"
#include "macro.cuh"
#include <cstdint>

namespace refactor::kernel::cuda {

    __global__ static void splitKernel(
        unsigned long long n,
        uint8_t const *data, unsigned int const *segments, uint8_t **outputs,
        unsigned int outputCount,
        unsigned int sum,
        unsigned int sub) {
        extern __shared__ uint8_t *shared[];
        auto outputs_ = shared;
        auto segments_ = reinterpret_cast<unsigned int *>(shared + outputCount);
        for (auto i = threadIdx.x; i < outputCount; i += blockDim.x) {
            outputs_[i] = outputs[i];
            segments_[i] = segments[i];
        }
        __syncthreads();
        for (auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                  step = blockDim.x * gridDim.x;
             tid < n;
             tid += step) {
            auto i = tid % sum, j = i * sub, k = 0u;
            while (j >= segments_[k]) { j -= segments_[k++]; }
            optimizedMemcpy(outputs_[k] + (tid / sum) * segments_[k] + j, data + tid * sub, sub);
        }
    }

    void launchSplit(
        KernelLaunchParameters const &params,
        void const *data, unsigned int const *segments, void **outputs,
        unsigned int outputCount,
        unsigned int sum,
        unsigned int sub) {
        splitKernel<<<
            params.gridSize,
            params.blockSize,
            outputCount *(sizeof(unsigned int) + sizeof(void *)),
            reinterpret_cast<hipStream_t>(params.stream)>>>(
            params.n,
            reinterpret_cast<uint8_t const *>(data),
            segments,
            reinterpret_cast<uint8_t **>(outputs),
            outputCount,
            sum,
            sub);
    }

}// namespace refactor::kernel::cuda
