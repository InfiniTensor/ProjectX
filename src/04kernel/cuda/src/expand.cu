#include "hip/hip_runtime.h"
﻿#include "kernel/cuda/expand.cuh"
#include "macro.cuh"
#include <cstdint>

namespace refactor::kernel::cuda {

    __global__ static void expandKernel(
        unsigned long long n,
        uint8_t const *__restrict__ data,
        expand::DimStride const *__restrict__ strides,
        uint8_t *__restrict__ output,
        unsigned int rank,
        unsigned int eleSize) {
        for (auto tid = blockIdx.x * blockDim.x + threadIdx.x,
                  step = blockDim.x * gridDim.x;
             tid < n;
             tid += step) {
            long rem = tid, i = 0;
            for (auto j = 0; j < rank; ++j) {
                auto o_ = __ldg(&(strides[j].o));
                auto i_ = __ldg(&(strides[j].i));
                i += rem / o_ * i_;
                rem %= o_;
            }
            optimizedMemcpy(output + tid * eleSize, data + i * eleSize, eleSize);
        }
    }

    void launchExpand(
        KernelLaunchParameters const &params,
        void const *data, expand::DimStride const *strides, void *output,
        unsigned int rank,
        unsigned int eleSize) {
        if (rank) {
            expandKernel<<<
                params.gridSize,
                params.blockSize,
                0,
                reinterpret_cast<hipStream_t>(params.stream)>>>(
                params.n,
                reinterpret_cast<uint8_t const *>(data),
                strides,
                reinterpret_cast<uint8_t *>(output),
                rank,
                eleSize);
        } else if (data != output) {
            hipMemcpyAsync(
                output,
                data,
                params.n * eleSize,
                hipMemcpyDeviceToDevice,
                reinterpret_cast<hipStream_t>(params.stream));
        }
    }

}// namespace refactor::kernel::cuda
