﻿#include "kernel/cuda/functions.cuh"
#include <cstdio>

namespace refactor::kernel::cuda {

    void sync() {
        auto state = hipDeviceSynchronize();
        if (state != hipSuccess) {
            printf("hipDeviceSynchronize failed: %s\n", hipGetErrorString(state));
            exit(1);
        }
    }

    void copyOut(void *dst, const void *src, size_t size) {
        sync();
        hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    }

    void setCudaDevice(int id) {
        hipSetDevice(id);
    }

}// namespace refactor::kernel::cuda
