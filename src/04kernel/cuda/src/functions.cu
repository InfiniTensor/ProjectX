﻿#include "kernel/cuda/functions.cuh"

namespace refactor::kernel::cuda {

    void sync() {
        hipDeviceSynchronize();
    }

}// namespace refactor::kernel::cuda
