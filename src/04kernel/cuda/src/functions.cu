﻿#include "kernel/cuda/functions.cuh"
#include "macro.cuh"
#include <cstdio>

namespace refactor::kernel::cuda {

    int currentDevice() {
        int device;
        CUDA_ASSERT(hipGetDevice(&device));
        return device;
    }

    void sync() {
        CUDA_ASSERT(hipDeviceSynchronize());
    }

    void copyOut(void *dst, const void *src, size_t size) {
        sync();
        CUDA_ASSERT(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
    }

    void setCudaDevice(int id) {
        hipSetDevice(id);
    }

}// namespace refactor::kernel::cuda
