﻿#include "cuda_mem.h"
#include <hip/hip_runtime.h>

namespace refactor::kernel::cuda {

    void *malloc(size_t bytes) {
        void *ans;
        hipMalloc(&ans, bytes);
        return ans;
    }
    void free(void *ptr) {
        hipFree(ptr);
    }
    void *memcpy_h2d(void *dst, void const *src, size_t bytes) noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyHostToDevice);
        return dst;
    }
    void *memcpy_d2h(void *dst, void const *src, size_t bytes) noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyDeviceToHost);
        return dst;
    }
    void *memcpy_d2d(void *dst, void const *src, size_t bytes) noexcept {
        hipMemcpy(dst, src, bytes, hipMemcpyDeviceToDevice);
        return dst;
    }

}// namespace refactor::kernel::cuda
